#include "hip/hip_runtime.h"
#include "kernels.cuh"

#include <hip/hip_runtime.h>

#include "utils.cuh"

__device__ __host__ fp_t gelu(fp_t x) {
    return 0.5f * x * (1.0f + tanhf(0.7978845608f * (x + 0.044715f * x * x * x)));
}

__global__ void embedding_kernel(
        id_t* token_ids,
        fp_t* wte,
        fp_t* wpe,
        fp_t* embeddings,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t n_embd) {
    // Calculate thread ID
    uint32_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }
    
    // Get token ID for current position
    id_t token_id = token_ids[idx_batch * seq_length + idx_seq];

    // Calculate offsets
    uint64_t offset_out = ((uint64_t)idx_batch * seq_length + idx_seq) * n_embd;
    uint64_t offset_wte = (uint64_t)token_id * n_embd;
    uint64_t offset_wpe = (uint64_t)idx_seq * n_embd;

    // Perform embedding lookup
    for (uint32_t i = 0; i < n_embd; i++) {
        embeddings[offset_out + i] = wte[offset_wte + i] + wpe[offset_wpe + i];
    }
}

__global__ void layer_normalization_kernel(
        fp_t* input,
        fp_t* gamma,
        fp_t* beta,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t n_embd) {
    // Calculate thread ID
    uint32_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Get the starting index for the current token
    uint64_t offset_input = ((uint64_t)idx_batch * seq_length + idx_seq) * n_embd;

    // Calculate mean
    fp_t mean = 0.0f;
    for (uint32_t i = 0; i < n_embd; i++) {
        mean += input[offset_input + i];
    }
    mean /= n_embd;

    // Calculate variance
    fp_t var = 0.0f;
    for (uint32_t i = 0; i < n_embd; i++) {
        fp_t diff = input[offset_input + i] - mean;
        var += diff * diff;
    }
    var /= n_embd;

    // Normalize and scale
    const fp_t epsilon = 1e-5f;
    fp_t inv_std = rsqrtf(var + epsilon);

    for (uint32_t i = 0; i < n_embd; i++) {
        fp_t normalized = (input[offset_input + i] - mean) * inv_std;
        input[offset_input + i] = gamma[i] * normalized + beta[i];
    }
}

__global__ void qkv_projection_kernel(
        fp_t* input,
        fp_t* output,
        fp_t* w_qkv,
        fp_t* b_qkv,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t n_embd) {
    // Calculate thread ID
    uint32_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Get the starting index for the current token
    uint64_t offset_input  = ((uint64_t) idx_batch * seq_length + idx_seq) * n_embd;
    uint64_t offset_output = ((uint64_t) idx_batch * seq_length + idx_seq) * (3 * n_embd);
    uint32_t qkv_size = 3 * n_embd; // Size of Q, K, V for each token

    // Perform QKV projection
    for (uint32_t i = 0; i < qkv_size; i++) {
        fp_t val = b_qkv[i];
        for (uint32_t j = 0; j < n_embd; j++) {
            val += input[offset_input + j] * w_qkv[j * qkv_size + i];
        }
        output[offset_output + i] = val;
    }
}

__global__ void multi_head_attention_kernel(
        fp_t* qkv,
        fp_t* output,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t n_head,
        uint32_t n_embd) {
    // Calculate thread ID
    uint32_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Calculate dimensions
    uint32_t d_k = n_embd / n_head; // Dimension of each head
    uint32_t qkv_size = 3 * n_embd; // Size of Q, K, V for each token

    // Scores register
    fp_t scores[SEQ_LENGTH_MAX];

    fp_t scale = rsqrtf(d_k);

    // Process each attention head
    for (uint32_t i_head = 0; i_head < n_head; i_head++) {
        // Calculate attention scores between current token and all other tokens
        fp_t max_val = -INFINITY;
        for (uint32_t j_token = 0; j_token < seq_length; j_token++) {
            // Causal masking: only attend to positions j_token <= i_token
            if (j_token <= idx_seq) {
                // Compute dot product
                fp_t dot = 0.0f;
                for (uint32_t d = 0; d < d_k; d++) {
                    // Q values for token idx_seq, head i_head
                    // K values for token j_token, head i_head
                    dot += qkv[idx_batch * seq_length * qkv_size + idx_seq * qkv_size + i_head * d_k + d] *
                           qkv[idx_batch * seq_length * qkv_size + j_token * qkv_size + 1 * n_embd + i_head * d_k + d];
                }
                scores[j_token] = dot * scale;
                max_val = fmaxf(max_val, scores[j_token]);
            }
        }

        // Softmax calculation for attention weights
        fp_t sum = 0.0f;
        for (uint32_t j_token = 0; j_token < seq_length; j_token++) {
            // Causal masking: masked tokens have zero weight
            if (j_token <= idx_seq) {
                scores[j_token] = expf(scores[j_token] - max_val);
                sum += scores[j_token];
            } else {
                scores[j_token] = 0.0f;
            }
        }

        for (uint32_t j_token = 0; j_token < seq_length; j_token++) {
            scores[j_token] /= sum;
        }

        // Calculate weighted sum of values
        for (uint32_t d = 0; d < d_k; d++) {
            fp_t weighted_sum = 0.0f;
            for (uint32_t j_token = 0; j_token < seq_length; j_token++) {
                // Get V values for token j_token, head i_head
                weighted_sum += scores[j_token] *
                                qkv[idx_batch * seq_length * qkv_size + j_token * qkv_size + 2 * n_embd + i_head * d_k + d];
            }
            // Use input as a temporary buffer to store head outputs
            output[idx_batch * seq_length * n_embd + idx_seq * n_embd + i_head * d_k + d] = weighted_sum;
        }
    }
}

__global__ void final_projection_kernel(
        fp_t* input,
        fp_t* output,
        fp_t* w_proj,
        fp_t* b_proj,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t n_embd) {
    // Calculate thread ID
    uint32_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Get the starting index for the current token
    uint64_t offset_input = ((uint64_t)idx_batch * seq_length + idx_seq) * n_embd;

    // Perform final projection
    for (uint32_t i = 0; i < n_embd; i++) {
        fp_t val = b_proj[i];
        for (uint32_t j = 0; j < n_embd; j++) {
            val += input[offset_input + j] * w_proj[j * n_embd + i];
        }
        output[offset_input + i] = val;
    }
}

__global__ void add_residual_kernel(
        fp_t* input,
        fp_t* residual,
        fp_t* output,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t n_embd) {
    // Calculate thread ID
    uint32_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Get the starting index for the current token
    uint64_t offset = ((uint64_t)idx_batch * seq_length + idx_seq) * n_embd;

    // Add residual connection
    for (uint32_t i = 0; i < n_embd; i++) {
        output[offset + i] = input[offset + i] + residual[offset + i];
    }
}

__global__ void mlp_kernel(
        fp_t* input,
        fp_t* output,
        fp_t* w_fc,
        fp_t* b_fc, 
        fp_t* w_proj,
        fp_t* b_proj,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t n_embd) {
    // Calculate thread ID
    uint32_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Intermediate register
    uint32_t intermediate_size = 4 * n_embd;
    fp_t intermediate[INTERMEDIATE_SIZE_MAX];

    // Get the starting index for the current token
    uint64_t offset_input = ((uint64_t)idx_batch * seq_length + idx_seq) * n_embd;

    // Compute feedforward layer
    for (uint32_t i = 0; i < intermediate_size; i++) {
        fp_t val = b_fc[i];
        for (uint32_t j = 0; j < n_embd; j++) {
            val += input[offset_input + j] * w_fc[j * intermediate_size + i];
        }
        intermediate[i] = gelu(val);
    }

    // Compute projection back to hidden size
    for (uint32_t i = 0; i < n_embd; i++) {
        fp_t val = b_proj[i];
        for (uint32_t j = 0; j < intermediate_size; j++) {
            val += intermediate[j] * w_proj[j * n_embd + i];
        }
        output[offset_input + i] = val;
    }
}

__global__ void lm_head_kernel(
        fp_t* hidden_state,
        fp_t* logits,
        fp_t* weights,
        fp_t* biases,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t n_vocab,
        uint32_t n_embd) {
    // Calculate thread ID
    uint32_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idx_vocab = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_vocab >= n_vocab) {
        return;
    }

    // Calculate output index
    uint64_t idx_out = ((uint64_t)idx_batch * n_vocab + idx_vocab);

    // Get the starting index for the current token
    uint64_t offset_input =  ((uint64_t)idx_batch * seq_length + (seq_length - 1)) * n_embd;
    uint64_t offset_weights = (uint64_t)idx_vocab * n_embd;

    // Compute logits
    logits[idx_out] = biases ? biases[idx_vocab] : 0.0f;
    for (uint32_t i = 0; i < n_embd; i++) {
        logits[idx_out] += hidden_state[offset_input + i] * weights[offset_weights + i];
    }
}