#include "hip/hip_runtime.h"
#include "kernels.cuh"

#include <hip/hip_runtime.h>

#include "utils.cuh"

__device__ __host__ fp_t gelu(fp_t x) {
    return 0.5f * x * (1.0f + tanhf(0.7978845608f * (x + 0.044715f * x * x * x)));
}

__global__ void embedding_kernel(
        id_t* token_ids,
        fp_t* wte,
        fp_t* wpe,
        fp_t* embeddings,
        uint64_t batch_size,
        uint64_t seq_length,
        uint64_t n_embd) {
    // Calculate thread ID
    uint64_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }
    
    // Get token ID for current position
    id_t token_id = token_ids[idx_batch * seq_length + idx_seq];

    // Calculate offsets
    uint64_t offset_out = (idx_batch * seq_length + idx_seq) * n_embd;
    uint64_t offset_wte = token_id * n_embd;
    uint64_t offset_wpe = idx_seq * n_embd;

    // Perform embedding lookup
    for (uint64_t i = 0; i < n_embd; i++) {
        embeddings[offset_out + i] = wte[offset_wte + i] + wpe[offset_wpe + i];
    }
}

__global__ void qkv_projection_kernel(
        fp_t* input,
        fp_t* output,
        fp_t* w_qkv,
        fp_t* b_qkv,
        uint64_t batch_size,
        uint64_t seq_length,
        uint64_t n_embd) {
    // Calculate thread ID
    uint64_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Get the starting index for the current token
    uint64_t offset_input = (idx_batch * seq_length + idx_seq) * n_embd;
    uint64_t offset_output = (idx_batch * seq_length + idx_seq) * (3 * n_embd);
    uint64_t qkv_size = 3 * n_embd; // Size of Q, K, V for each token

    // Perform QKV projection
    for (uint64_t i = 0; i < qkv_size; i++) {
        fp_t val = b_qkv[i];
        for (uint64_t j = 0; j < n_embd; j++) {
            val += input[offset_input + j] * w_qkv[j * qkv_size + i];
        }
        output[offset_output + i] = val;
    }
}

__global__ void layer_normalization_kernel(
        fp_t* input,
        fp_t* gamma,
        fp_t* beta,
        uint64_t batch_size,
        uint64_t seq_length,
        uint64_t n_embd) {
    // Calculate thread ID
    uint64_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Get the starting index for the current token
    uint64_t offset_input = (idx_batch * seq_length + idx_seq) * n_embd;

    // Calculate mean
    fp_t mean = 0.0f;
    for (uint64_t i = 0; i < n_embd; i++) {
        mean += input[offset_input + i];
    }
    mean /= n_embd;

    // Calculate variance
    fp_t var = 0.0f;
    for (uint64_t i = 0; i < n_embd; i++) {
        fp_t diff = input[offset_input + i] - mean;
        var += diff * diff;
    }
    var /= n_embd;

    // Normalize and scale
    const fp_t epsilon = 1e-5f;
    fp_t inv_std = rsqrtf(var + epsilon);

    for (uint64_t i = 0; i < n_embd; i++) {
        fp_t normalized = (input[offset_input + i] - mean) * inv_std;
        input[offset_input + i] = gamma[i] * normalized + beta[i];
    }
}

__global__ void multi_head_attention_kernel(
        fp_t* qkv,
        fp_t* output,
        uint64_t batch_size,
        uint64_t seq_length,
        uint64_t n_head,
        uint64_t n_embd) {
    // Calculate thread ID
    uint64_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Calculate dimensions
    uint64_t d_k = n_embd / n_head; // Dimension of each head
    uint64_t qkv_size = 3 * n_embd; // Size of Q, K, V for each token

    // Scores register
    fp_t scores[SEQ_LENGTH_MAX];

    fp_t scale = rsqrtf(d_k);

    // Process each attention head
    for (uint64_t i_head = 0; i_head < n_head; i_head++) {
        // Calculate attention scores between current token and all other tokens
        fp_t max_val = -INFINITY;
        for (uint64_t j_token = 0; j_token < seq_length; j_token++) {
            // Causal masking: only attend to positions j_token <= i_token
            if (j_token <= idx_seq) {
                // Compute dot product
                fp_t dot = 0.0f;
                for (uint64_t d = 0; d < d_k; d++) {
                    // Q values for token idx_seq, head i_head
                    // K values for token j_token, head i_head
                    dot += qkv[idx_batch * seq_length * qkv_size + idx_seq * qkv_size + i_head * d_k + d] *
                           qkv[idx_batch * seq_length * qkv_size + j_token * qkv_size + 1 * n_embd + i_head * d_k + d];
                }
                scores[j_token] = dot * scale;
                max_val = fmaxf(max_val, scores[j_token]);
            }
        }

        // Softmax calculation for attention weights
        fp_t sum = 0.0f;
        for (uint64_t j_token = 0; j_token < seq_length; j_token++) {
            // Causal masking: masked tokens have zero weight
            if (j_token <= idx_seq) {
                scores[j_token] = expf(scores[j_token] - max_val);
                sum += scores[j_token];
            } else {
                scores[j_token] = 0.0f;
            }
        }

        for (uint64_t j_token = 0; j_token < seq_length; j_token++) {
            scores[j_token] /= sum;
        }

        // Calculate weighted sum of values
        for (uint64_t d = 0; d < d_k; d++) {
            fp_t weighted_sum = 0.0f;
            for (uint64_t j_token = 0; j_token < seq_length; j_token++) {
                // Get V values for token j_token, head i_head
                weighted_sum += scores[j_token] *
                                qkv[idx_batch * seq_length * qkv_size + j_token * qkv_size + 2 * n_embd + i_head * d_k + d];
            }
            // Use input as a temporary buffer to store head outputs
            output[idx_batch * seq_length * n_embd + idx_seq * n_embd + i_head * d_k + d] = weighted_sum;
        }
    }
}

__global__ void final_projection_kernel(
        fp_t* input,
        fp_t* output,
        fp_t* w_proj,
        fp_t* b_proj,
        uint64_t batch_size,
        uint64_t seq_length,
        uint64_t n_embd) {
    // Calculate thread ID
    uint64_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Get the starting index for the current token
    uint64_t offset_input = (idx_batch * seq_length + idx_seq) * n_embd;

    // Perform final projection
    for (uint64_t i = 0; i < n_embd; i++) {
        fp_t val = b_proj[i];
        for (uint64_t j = 0; j < n_embd; j++) {
            val += input[offset_input + j] * w_proj[j * n_embd + i];
        }
        output[offset_input + i] = val;
    }
}

__global__ void add_residual_kernel(
        fp_t* input,
        fp_t* residual,
        fp_t* output,
        uint64_t batch_size,
        uint64_t seq_length,
        uint64_t n_embd) {
    // Calculate thread ID
    uint64_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Get the starting index for the current token
    uint64_t offset = (idx_batch * seq_length + idx_seq) * n_embd;

    // Add residual connection
    for (uint64_t i = 0; i < n_embd; i++) {
        output[offset + i] = input[offset + i] + residual[offset + i];
    }
}

__global__ void mlp_kernel(
        fp_t* input,
        fp_t* output,
        fp_t* w_fc,
        fp_t* b_fc, 
        fp_t* w_proj,
        fp_t* b_proj,
        uint64_t batch_size,
        uint64_t seq_length,
        uint64_t n_embd) {
    // Calculate thread ID
    uint64_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Intermediate register
    uint64_t intermediate_size = 4 * n_embd;
    fp_t intermediate[INTERMEDIATE_SIZE_MAX];

    // Get the starting index for the current token
    uint64_t offset_input = (idx_batch * seq_length + idx_seq) * n_embd;

    // Compute feedforward layer
    for (uint64_t i = 0; i < intermediate_size; i++) {
        fp_t val = b_fc[i];
        for (uint64_t j = 0; j < n_embd; j++) {
            val += input[offset_input + j] * w_fc[j * intermediate_size + i];
        }
        intermediate[i] = gelu(val);
    }

    // Compute projection back to hidden size
    for (uint64_t i = 0; i < n_embd; i++) {
        fp_t val = b_proj[i];
        for (uint64_t j = 0; j < intermediate_size; j++) {
            val += intermediate[j] * w_proj[j * n_embd + i];
        }
        output[offset_input + i] = val;
    }
}

__global__ void lm_head_kernel(
        fp_t* hidden_state,
        fp_t* logits,
        fp_t* weights,
        fp_t* biases,
        uint64_t batch_size,
        uint64_t seq_length,
        uint64_t n_vocab,
        uint64_t n_embd) {
    // Calculate thread ID
    uint64_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t idx_vocab = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_vocab >= n_vocab) {
        return;
    }

    // Calculate output index
    uint64_t idx_out = (idx_batch * n_vocab + idx_vocab);

    // Get the starting index for the current token
    uint64_t offset_input =  (idx_batch * seq_length + (seq_length - 1)) * n_embd;
    uint64_t offset_weights = idx_vocab * n_embd;

    // Compute logits
    logits[idx_out] = biases ? biases[idx_vocab] : 0.0f;
    for (uint64_t i = 0; i < n_embd; i++) {
        logits[idx_out] += hidden_state[offset_input + i] * weights[offset_weights + i];
    }
}