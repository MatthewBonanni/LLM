#include "hip/hip_runtime.h"
#include "kernels.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#include "utils.cuh"

using namespace nvcuda;

__device__ __host__ fp_t gelu(fp_t x) {
    return 0.5f * x * (1.0f + tanhf(0.7978845608f * (x + 0.044715f * x * x * x)));
}

template <uint32_t BLOCK_SIZE>
__global__ void embedding_kernel(
        const id_t* __restrict__ token_ids,
        const fp_t* __restrict__ wte,
        const fp_t* __restrict__ wpe,
        fp_t* __restrict__ embeddings,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t seq_offset,
        uint32_t n_embd) {
    // Calculate thread ID
    const uint32_t idx_batch = blockIdx.x;
    const uint32_t idx_seq = blockIdx.y;
    const uint32_t tidx = threadIdx.x;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }
    
    // Shared memory for token ID (one per block)
    __shared__ id_t token_id;
    if (tidx == 0) {
        token_id = token_ids[idx_batch * seq_length + idx_seq];
    }
    __syncthreads();
    
    // Calculate base offsets
    const uint64_t out_offset = ((uint64_t)idx_batch * seq_length + idx_seq) * n_embd;
    const uint64_t wte_offset = (uint64_t)token_id * n_embd;
    const uint64_t wpe_offset = ((uint64_t)idx_seq + seq_offset) * n_embd;
    
    // Iterate over the embedding dimension in chunks of 8
    #pragma unroll
    for (uint32_t i = tidx * 8; i < n_embd; i += BLOCK_SIZE * 8) {
        // Load first 4 elements
        const float4 wte_vec1 = *reinterpret_cast<const float4*>(&wte[wte_offset + i]);
        const float4 wpe_vec1 = *reinterpret_cast<const float4*>(&wpe[wpe_offset + i]);
        
        // Load next 4 elements
        const float4 wte_vec2 = *reinterpret_cast<const float4*>(&wte[wte_offset + i + 4]);
        const float4 wpe_vec2 = *reinterpret_cast<const float4*>(&wpe[wpe_offset + i + 4]);
        
        // Store first 4 elements
        *reinterpret_cast<float4*>(&embeddings[out_offset + i]) = make_float4(
            wte_vec1.x + wpe_vec1.x,
            wte_vec1.y + wpe_vec1.y,
            wte_vec1.z + wpe_vec1.z,
            wte_vec1.w + wpe_vec1.w
        );
        
        // Store next 4 elements
        *reinterpret_cast<float4*>(&embeddings[out_offset + i + 4]) = make_float4(
            wte_vec2.x + wpe_vec2.x,
            wte_vec2.y + wpe_vec2.y,
            wte_vec2.z + wpe_vec2.z,
            wte_vec2.w + wpe_vec2.w
        );
    }
}

// Explicit instantiation
template __global__ void embedding_kernel<128>(
        const id_t* __restrict__ token_ids,
        const fp_t* __restrict__ wte,
        const fp_t* __restrict__ wpe,
        fp_t* __restrict__ embeddings,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t seq_offset,
        uint32_t n_embd);

template <uint32_t BLOCK_SIZE, uint32_t WARPS_PER_BLOCK>
__global__ void layer_normalization_kernel(
        fp_t* __restrict__ input,
        const fp_t* __restrict__ gamma,
        const fp_t* __restrict__ beta,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t n_embd) {
    // Calculate thread ID
    const uint32_t idx_batch = blockIdx.x;
    const uint32_t idx_seq = blockIdx.y;
    const uint32_t tidx = threadIdx.x;
    const uint32_t lane_id = tidx % WARP_SIZE;
    const uint32_t warp_id = tidx / WARP_SIZE;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Get the starting index for the current token
    const uint64_t offset_input = ((uint64_t)idx_batch * seq_length + idx_seq) * n_embd;
    
    // Shared memory for partial sums - organized by warp for efficient access
    __shared__ fp_t s_mean[WARPS_PER_BLOCK];
    __shared__ fp_t s_variance[WARPS_PER_BLOCK];
    
    // Local accumulators
    fp_t sum = 0.0f;
    fp_t sq_sum = 0.0f;
    
    // Calculate local sum and squared sum (with coalesced memory access)
    #pragma unroll
    for (uint32_t i = tidx * 4; i < n_embd; i += BLOCK_SIZE * 4) {
        float4 val4 = *reinterpret_cast<float4*>(&input[offset_input + i]);
        sum += val4.x + val4.y + val4.z + val4.w;
        sq_sum += val4.x * val4.x +
                  val4.y * val4.y +
                  val4.z * val4.z +
                  val4.w * val4.w;
    }
    
    // Warp-level reduction using shuffle operations
    #pragma unroll
    for (uint32_t offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        sum    += __shfl_down_sync(0xffffffff, sum,    offset);
        sq_sum += __shfl_down_sync(0xffffffff, sq_sum, offset);
    }
    
    // First thread in each warp writes partial results
    if (lane_id == 0) {
        s_mean[warp_id] = sum;
        s_variance[warp_id] = sq_sum;
    }
    __syncthreads();
    
    // Final reduction across warps (done by first warp)
    if (warp_id == 0) {
        // Load 0 for lanes that would access out of bounds
        fp_t warp_sum = (lane_id < WARPS_PER_BLOCK) ? s_mean[lane_id] : 0.0f;
        fp_t warp_sq_sum = (lane_id < WARPS_PER_BLOCK) ? s_variance[lane_id] : 0.0f;
        
        // Warp-level reduction again
        #pragma unroll
        for (uint32_t offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
            warp_sum += __shfl_down_sync(0xffffffff, warp_sum, offset);
            warp_sq_sum += __shfl_down_sync(0xffffffff, warp_sq_sum, offset);
        }
        
        // First thread calculates final values
        if (lane_id == 0) {
            const fp_t inv_n = 1.0f / n_embd;
            s_mean[0] = warp_sum * inv_n;
            fp_t variance = fmaxf(warp_sq_sum * inv_n - s_mean[0] * s_mean[0], 0.0f);
            s_variance[0] = rsqrtf(variance + 1e-5f);  // inverse standard deviation
        }
    }
    __syncthreads();
    
    // Load final mean and inv_std
    const fp_t mean = s_mean[0];
    const fp_t inv_std = s_variance[0];
    
    // Normalize and scale - ensure coalesced memory access
    // Each thread handles multiple sequential elements for better instruction throughput
    #pragma unroll
    for (uint32_t i = tidx * 4; i < n_embd; i += BLOCK_SIZE * 4) {
        float4 input_vec = *reinterpret_cast<float4*>(&input[offset_input + i]);
        float4 gamma_vec = *reinterpret_cast<const float4*>(&gamma[i]);
        float4 beta_vec = *reinterpret_cast<const float4*>(&beta[i]);

        input_vec.x = (input_vec.x - mean) * inv_std * gamma_vec.x + beta_vec.x;
        input_vec.y = (input_vec.y - mean) * inv_std * gamma_vec.y + beta_vec.y;
        input_vec.z = (input_vec.z - mean) * inv_std * gamma_vec.z + beta_vec.z;
        input_vec.w = (input_vec.w - mean) * inv_std * gamma_vec.w + beta_vec.w;

        *reinterpret_cast<float4*>(&input[offset_input + i]) = input_vec;
    }
}

// Explicit instantiation
template __global__ void layer_normalization_kernel<256, 8>(
        fp_t* __restrict__ input,
        const fp_t* __restrict__ gamma,
        const fp_t* __restrict__ beta,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t n_embd);

__global__ void q_projection_kernel(
        const fp_t* __restrict__ hidden_states,
        fp_t* __restrict__ q,
        const fp_t* __restrict__ w_qkv,
        const fp_t* __restrict__ b_qkv,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t n_embd) {
    // Calculate batch and embedding indices
    uint32_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idx_embd = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_embd  >= n_embd) {
        return;
    }

    // Calculate memory offsets
    uint64_t hidden_offset = ((uint64_t)idx_batch * seq_length + (seq_length - 1)) * n_embd;
    uint64_t q_offset = (uint64_t)idx_batch * n_embd;

    // Perform Q projection
    float val = b_qkv[idx_embd];
    for (uint32_t i = 0; i < n_embd; i++) {
        val += hidden_states[hidden_offset + i] * w_qkv[i * 3 * n_embd + idx_embd];
    }
    q[q_offset + idx_embd] = val;
}

__global__ void kv_projection_kernel(
        const fp_t* __restrict__ hidden_states,
        half* __restrict__ kv,
        const fp_t* __restrict__ w_qkv,
        const fp_t* __restrict__ b_qkv,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t seq_offset,
        uint32_t n_embd) {
    // Calculate batch and sequence indices
    uint32_t idx_batch = blockIdx.z;
    uint32_t idx_seq = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Calculate tile indices
    uint32_t tile_x = threadIdx.x;
    uint32_t tile_y = threadIdx.y % WMMA_M;

    // Calculate memory offsets
    uint64_t hidden_offset = ((uint64_t)idx_batch * seq_length + idx_seq) * n_embd;
    uint64_t kv_offset = ((uint64_t)idx_batch * (seq_length + seq_offset) + idx_seq + seq_offset) * (2 * n_embd);
    
    // Allocate shared memory for tiling - optimized layout for KV
    extern __shared__ half smem[];
    half* hidden_shared = &smem[0];
    half* k_weights_shared = &smem[WMMA_M * n_embd];
    half* v_weights_shared = &smem[WMMA_M * n_embd + WMMA_M * WMMA_K];
    
    // Initialize tensor core fragments
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> hidden_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> k_weights_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> v_weights_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> k_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> v_frag;
    
    wmma::fill_fragment(hidden_frag, 0.0f);
    wmma::fill_fragment(k_weights_frag, 0.0f);
    wmma::fill_fragment(v_weights_frag, 0.0f);
    wmma::fill_fragment(k_frag, 0.0f);
    wmma::fill_fragment(v_frag, 0.0f);
    
    // Copy bias values into output arrays
    for (uint32_t i = tile_x; i < n_embd; i += blockDim.x) {
        kv[kv_offset +          i] = __float2half(b_qkv[    n_embd + i]);
        kv[kv_offset + n_embd + i] = __float2half(b_qkv[2 * n_embd + i]);
    }
    
    __syncthreads();
    
    // Process in tiles of WMMA_K
    for (uint32_t k_idx = 0; k_idx < n_embd; k_idx += WMMA_K) {
        // Load hidden states into shared memory
        for (uint32_t i = tile_x; i < WMMA_K && (k_idx + i) < n_embd; i += blockDim.x) {
            hidden_shared[tile_y * WMMA_K + i] = __float2half(hidden_states[hidden_offset + k_idx + i]);
        }
        
        // Load KV weights into shared memory
        for (uint32_t i = tile_y; i < WMMA_M && i < n_embd; i += blockDim.y) {
            for (uint32_t j = tile_x; j < WMMA_K && (k_idx + j) < n_embd; j += blockDim.x) {
                k_weights_shared[i * WMMA_K + j] = __float2half(w_qkv[(k_idx + j) * (3 * n_embd) + n_embd + i]);
                v_weights_shared[i * WMMA_K + j] = __float2half(w_qkv[(k_idx + j) * (3 * n_embd) + (2 * n_embd) + i]);
            }
        }
        
        __syncthreads();
        
        // Load fragments
        wmma::load_matrix_sync(hidden_frag, hidden_shared, WMMA_K);
        wmma::load_matrix_sync(k_weights_frag, k_weights_shared, WMMA_K);
        wmma::load_matrix_sync(v_weights_frag, v_weights_shared, WMMA_K);
        
        // Perform matrix multiplication using tensor cores
        wmma::mma_sync(k_frag, hidden_frag, k_weights_frag, k_frag);
        wmma::mma_sync(v_frag, hidden_frag, v_weights_frag, v_frag);
        
        // Synchronize before next iteration
        __syncthreads();
    }
    
    // Store results
    uint32_t warp_id = threadIdx.y / warpSize;
    uint32_t lane_id = threadIdx.x + (threadIdx.y % warpSize) * blockDim.x;
    if (lane_id < WMMA_M * WMMA_N) {
        uint32_t col = lane_id % WMMA_N;
        uint32_t row = lane_id / WMMA_N;
        if (row < WMMA_M && col < WMMA_N && (row * WMMA_N + col) < n_embd) {
            uint32_t k_idx = kv_offset + (warp_id * WMMA_M * WMMA_N) + row * WMMA_N + col;
            uint32_t v_idx = k_idx + n_embd;
            if (k_idx < kv_offset + n_embd) {
                atomicAdd(&kv[k_idx], __float2half(k_frag.x[row * WMMA_N + col]));
                atomicAdd(&kv[v_idx], __float2half(v_frag.x[row * WMMA_N + col]));
            }
        }
    }
}

__global__ void multi_head_attention_kernel(
        const fp_t* __restrict__ q,
        const half* __restrict__ kv,
        fp_t* __restrict__ output,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t seq_offset,
        uint32_t n_head,
        uint32_t n_embd) {
    // Calculate thread ID
    uint32_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Calculate dimensions
    uint32_t d_k = n_embd / n_head; // Dimension of each head
    uint32_t kv_size = 2 * n_embd; // Size of Q, K, V for each token

    // Scores register
    fp_t scores[SEQ_LENGTH_MAX];

    fp_t scale = rsqrtf(d_k);

    // Process each attention head
    for (uint32_t i_head = 0; i_head < n_head; i_head++) {
        // Calculate attention scores between current token and all other tokens
        fp_t max_val = -INFINITY;
        for (uint32_t j_token = 0; j_token < (seq_length + seq_offset); j_token++) {
            // Causal masking: only attend to positions j_token <= i_token
            if (j_token <= idx_seq) {
                // Compute dot product
                fp_t dot = 0.0f;
                for (uint32_t d = 0; d < d_k; d++) {
                    // Q values for token idx_seq, head i_head
                    // K values for token j_token, head i_head
                    dot += q[idx_batch * seq_length * n_embd +
                             idx_seq * n_embd +
                             i_head * d_k +
                             d] *
                           __half2float(kv[idx_batch * (seq_length + seq_offset) * kv_size +
                                           j_token * kv_size +
                                           0 * n_embd +
                                           i_head * d_k +
                                           d]);
                }
                scores[j_token] = dot * scale;
                max_val = fmaxf(max_val, scores[j_token]);
            }
        }

        // Softmax calculation for attention weights
        fp_t sum = 0.0f;
        for (uint32_t j_token = 0; j_token < seq_length; j_token++) {
            // Causal masking: masked tokens have zero weight
            if (j_token <= idx_seq) {
                scores[j_token] = expf(scores[j_token] - max_val);
                sum += scores[j_token];
            } else {
                scores[j_token] = 0.0f;
            }
        }

        for (uint32_t j_token = 0; j_token < seq_length; j_token++) {
            scores[j_token] /= sum;
        }

        // Calculate weighted sum of values
        for (uint32_t d = 0; d < d_k; d++) {
            fp_t weighted_sum = 0.0f;
            for (uint32_t j_token = 0; j_token < (seq_length + seq_offset); j_token++) {
                // Get V values for token j_token, head i_head
                weighted_sum += scores[j_token] *
                                __half2float(kv[idx_batch * (seq_length + seq_offset) * kv_size +
                                                j_token * kv_size +
                                                1 * n_embd +
                                                i_head * d_k +
                                                d]);
            }
            // Use input as a temporary buffer to store head outputs
            output[idx_batch * seq_length * n_embd +
                   idx_seq * n_embd +
                   i_head * d_k +
                   d] = weighted_sum;
        }
    }
}

__global__ void final_projection_kernel(
        const fp_t* __restrict__ input,
        fp_t* __restrict__ output,
        const fp_t* __restrict__ w_proj,
        const fp_t* __restrict__ b_proj,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t n_embd) {
    // Calculate thread ID
    uint32_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Get the starting index for the current token
    uint64_t offset_input = ((uint64_t)idx_batch * seq_length + idx_seq) * n_embd;

    // Perform final projection
    for (uint32_t i = 0; i < n_embd; i++) {
        fp_t val = b_proj[i];
        for (uint32_t j = 0; j < n_embd; j++) {
            val += input[offset_input + j] * w_proj[j * n_embd + i];
        }
        output[offset_input + i] = val;
    }
}

__global__ void add_residual_kernel(
        const fp_t* __restrict__ input,
        const fp_t* __restrict__ residual,
        fp_t* __restrict__ output,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t n_embd) {
    // Calculate thread ID
    uint32_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Get the starting index for the current token
    uint64_t offset = ((uint64_t)idx_batch * seq_length + idx_seq) * n_embd;

    // Add residual connection
    for (uint32_t i = 0; i < n_embd; i++) {
        output[offset + i] = input[offset + i] + residual[offset + i];
    }
}

__global__ void mlp_kernel(
        const fp_t* __restrict__ input,
        fp_t* __restrict__ output,
        const fp_t* __restrict__ w_fc,
        const fp_t* __restrict__ b_fc, 
        const fp_t* __restrict__ w_proj,
        const fp_t* __restrict__ b_proj,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t n_embd) {
    // Calculate thread ID
    uint32_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idx_seq   = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_seq   >= seq_length) {
        return;
    }

    // Intermediate register
    uint32_t intermediate_size = 4 * n_embd;
    fp_t intermediate[INTERMEDIATE_SIZE];

    // Get the starting index for the current token
    uint64_t offset_input = ((uint64_t)idx_batch * seq_length + idx_seq) * n_embd;

    // Compute feedforward layer
    for (uint32_t i = 0; i < intermediate_size; i++) {
        fp_t val = b_fc[i];
        for (uint32_t j = 0; j < n_embd; j++) {
            val += input[offset_input + j] * w_fc[j * intermediate_size + i];
        }
        intermediate[i] = gelu(val);
    }

    // Compute projection back to hidden size
    for (uint32_t i = 0; i < n_embd; i++) {
        fp_t val = b_proj[i];
        for (uint32_t j = 0; j < intermediate_size; j++) {
            val += intermediate[j] * w_proj[j * n_embd + i];
        }
        output[offset_input + i] = val;
    }
}

__global__ void lm_head_kernel(
        const fp_t* __restrict__ hidden_state,
        fp_t* __restrict__ logits,
        const fp_t* __restrict__ weights,
        const fp_t* __restrict__ biases,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t n_vocab,
        uint32_t n_embd) {
    // Calculate thread ID
    uint32_t idx_batch = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idx_vocab = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (idx_batch >= batch_size ||
        idx_vocab >= n_vocab) {
        return;
    }

    // Calculate output index
    uint64_t idx_out = ((uint64_t)idx_batch * n_vocab + idx_vocab);

    // Get the starting index for the current token
    uint64_t offset_input =  ((uint64_t)idx_batch * seq_length + (seq_length - 1)) * n_embd;
    uint64_t offset_weights = (uint64_t)idx_vocab * n_embd;

    // Compute logits
    logits[idx_out] = biases ? biases[idx_vocab] : 0.0f;
    for (uint32_t i = 0; i < n_embd; i++) {
        logits[idx_out] += hidden_state[offset_input + i] * weights[offset_weights + i];
    }
}