#include "layer.cuh"

#include <hip/hip_runtime.h>

#include <stdexcept>

#include "utils.cuh"
#include "io.cuh"
#include "kernels.cuh"

Layer::Layer(int n_embd, int n_head) : 
        n_embd(n_embd),
        n_head(n_head),
        d_attn_c_attn_w_0(nullptr),
        d_attn_c_attn_b_0(nullptr),
        d_attn_c_proj_w_0(nullptr),
        d_attn_c_proj_b_0(nullptr),
        d_ln_1_b_0(nullptr),
        d_ln_1_g_0(nullptr),
        d_ln_2_b_0(nullptr),
        d_ln_2_g_0(nullptr),
        d_mlp_c_fc_w_0(nullptr),
        d_mlp_c_fc_b_0(nullptr),
        d_mlp_c_proj_w_0(nullptr),
        d_mlp_c_proj_b_0(nullptr) {
    // Allocate memory on host
    h_attn_c_attn_w_0.resize(n_embd * 3 * n_embd);
    h_attn_c_attn_b_0.resize(3 * n_embd);
    h_attn_c_proj_w_0.resize(n_embd * n_embd);
    h_attn_c_proj_b_0.resize(n_embd);
    h_ln_1_b_0.resize(n_embd);
    h_ln_1_g_0.resize(n_embd);
    h_ln_2_b_0.resize(n_embd);
    h_ln_2_g_0.resize(n_embd);
    h_mlp_c_fc_w_0.resize(n_embd * 4 * n_embd);
    h_mlp_c_fc_b_0.resize(4 * n_embd);
    h_mlp_c_proj_w_0.resize(4 * n_embd * n_embd);
    h_mlp_c_proj_b_0.resize(n_embd);

    // Allocate memory on device
    CHECK_CUDA(hipMalloc(&d_attn_c_attn_w_0, n_embd * 3 * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_attn_c_attn_b_0, 3 * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_attn_c_proj_w_0, n_embd * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_attn_c_proj_b_0, n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_ln_1_b_0, n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_ln_1_g_0, n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_ln_2_b_0, n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_ln_2_g_0, n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_fc_w_0, n_embd * 4 * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_fc_b_0, 4 * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_proj_w_0, 4 * n_embd * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_proj_b_0, n_embd * sizeof(float)));
}

Layer::~Layer() {
    // Free memory on device
    CHECK_CUDA(hipFree(d_attn_c_attn_w_0));
    CHECK_CUDA(hipFree(d_attn_c_attn_b_0));
    CHECK_CUDA(hipFree(d_attn_c_proj_w_0));
    CHECK_CUDA(hipFree(d_attn_c_proj_b_0));
    CHECK_CUDA(hipFree(d_ln_1_b_0));
    CHECK_CUDA(hipFree(d_ln_1_g_0));
    CHECK_CUDA(hipFree(d_ln_2_b_0));
    CHECK_CUDA(hipFree(d_ln_2_g_0));
    CHECK_CUDA(hipFree(d_mlp_c_fc_w_0));
    CHECK_CUDA(hipFree(d_mlp_c_fc_b_0));
    CHECK_CUDA(hipFree(d_mlp_c_proj_w_0));
    CHECK_CUDA(hipFree(d_mlp_c_proj_b_0));
}

void Layer::apply(float* d_hidden_states, float* d_residual, float* d_temp, int seq_length) {
    // Calculate dimensions
    int block_size = 256; // Using a fixed block size that works well for most cases
    int grid_size = (seq_length * n_embd + block_size - 1) / block_size;

    // Allocate temporary buffers
    float* d_qkv = nullptr;
    CHECK_CUDA(hipMalloc(&d_qkv, seq_length * 3 * n_embd * sizeof(float)));
    
    // Step 1: Save input for residual connection
    CHECK_CUDA(hipMemcpy(d_residual, d_hidden_states, seq_length * n_embd * sizeof(float), hipMemcpyDeviceToDevice));
    
    // Step 2: First layer normalization
    layer_normalization_kernel<<<grid_size, block_size>>>(
        d_hidden_states, d_ln_1_g_0, d_ln_1_b_0, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());

    // Step 3: Multi-head attention
    // Step 3.1: QKV projection
    qkv_projection_kernel<<<grid_size, block_size>>>(
        d_hidden_states, d_qkv, 
        d_attn_c_attn_w_0, d_attn_c_attn_b_0, 
        seq_length, n_embd, 3 * n_embd);
    CHECK_CUDA(hipGetLastError());

    // Step 3.2: Multi-head attention
    multi_head_attention_kernel<<<grid_size, block_size>>>(
        d_qkv, d_hidden_states, 
        seq_length, n_embd, n_head);
    CHECK_CUDA(hipGetLastError());

    // Step 3.3: Final projection
    final_projection_kernel<<<grid_size, block_size>>>(
        d_hidden_states, d_temp,
        d_attn_c_proj_w_0, d_attn_c_proj_b_0,
        seq_length, n_embd);

    // Step 4: Add residual connection
    add_residual_kernel<<<grid_size, block_size>>>(
        d_temp, d_residual, d_hidden_states, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());

    // Step 5: Save output for residual connection
    CHECK_CUDA(hipMemcpy(d_residual, d_hidden_states, seq_length * n_embd * sizeof(float), hipMemcpyDeviceToDevice));

    // Step 6: Second layer normalization
    layer_normalization_kernel<<<grid_size, block_size>>>(
        d_hidden_states, d_ln_2_g_0, d_ln_2_b_0, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());

    // Step 7: MLP (feedforward network)
    mlp_kernel<<<grid_size, block_size>>>(
        d_hidden_states, d_temp,
        d_mlp_c_fc_w_0, d_mlp_c_fc_b_0,
        d_mlp_c_proj_w_0, d_mlp_c_proj_b_0,
        seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());

    // Step 8: Add residual connection
    add_residual_kernel<<<grid_size, block_size>>>(
        d_temp, d_residual, d_hidden_states, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());

    // Free temporary buffers
    CHECK_CUDA(hipFree(d_qkv));
}

void Layer::load_from_hdf5(hid_t file_id, const std::string& layer_path) {
    read_dataset(file_id, layer_path + "/attn/c_attn/w_0", h_attn_c_attn_w_0);
    read_dataset(file_id, layer_path + "/attn/c_attn/b_0", h_attn_c_attn_b_0);
    read_dataset(file_id, layer_path + "/attn/c_proj/w_0", h_attn_c_proj_w_0);
    read_dataset(file_id, layer_path + "/attn/c_proj/b_0", h_attn_c_proj_b_0);
    read_dataset(file_id, layer_path + "/ln_1/b_0",        h_ln_1_b_0);
    read_dataset(file_id, layer_path + "/ln_1/g_0",        h_ln_1_g_0);
    read_dataset(file_id, layer_path + "/ln_2/b_0",        h_ln_2_b_0);
    read_dataset(file_id, layer_path + "/ln_2/g_0",        h_ln_2_g_0);
    read_dataset(file_id, layer_path + "/mlp/c_fc/w_0",    h_mlp_c_fc_w_0);
    read_dataset(file_id, layer_path + "/mlp/c_fc/b_0",    h_mlp_c_fc_b_0);
    read_dataset(file_id, layer_path + "/mlp/c_proj/w_0",  h_mlp_c_proj_w_0);
    read_dataset(file_id, layer_path + "/mlp/c_proj/b_0",  h_mlp_c_proj_b_0);
}

void Layer::copy_host_to_device() {
    CHECK_CUDA(hipMemcpy(d_attn_c_attn_w_0, h_attn_c_attn_w_0.data(), h_attn_c_attn_w_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_attn_b_0, h_attn_c_attn_b_0.data(), h_attn_c_attn_b_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_proj_w_0, h_attn_c_proj_w_0.data(), h_attn_c_proj_w_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_proj_b_0, h_attn_c_proj_b_0.data(), h_attn_c_proj_b_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_1_b_0, h_ln_1_b_0.data(), h_ln_1_b_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_1_g_0, h_ln_1_g_0.data(), h_ln_1_g_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_2_b_0, h_ln_2_b_0.data(), h_ln_2_b_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_2_g_0, h_ln_2_g_0.data(), h_ln_2_g_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_fc_w_0, h_mlp_c_fc_w_0.data(), h_mlp_c_fc_w_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_fc_b_0, h_mlp_c_fc_b_0.data(), h_mlp_c_fc_b_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_proj_w_0, h_mlp_c_proj_w_0.data(), h_mlp_c_proj_w_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_proj_b_0, h_mlp_c_proj_b_0.data(), h_mlp_c_proj_b_0.size() * sizeof(float), hipMemcpyHostToDevice));
}

void Layer::copy_device_to_host() {
    CHECK_CUDA(hipMemcpy(h_attn_c_attn_w_0.data(), d_attn_c_attn_w_0, h_attn_c_attn_w_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_attn_c_attn_b_0.data(), d_attn_c_attn_b_0, h_attn_c_attn_b_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_attn_c_proj_w_0.data(), d_attn_c_proj_w_0, h_attn_c_proj_w_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_attn_c_proj_b_0.data(), d_attn_c_proj_b_0, h_attn_c_proj_b_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_ln_1_b_0.data(), d_ln_1_b_0, h_ln_1_b_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_ln_1_g_0.data(), d_ln_1_g_0, h_ln_1_g_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_ln_2_b_0.data(), d_ln_2_b_0, h_ln_2_b_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_ln_2_g_0.data(), d_ln_2_g_0, h_ln_2_g_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_mlp_c_fc_w_0.data(), d_mlp_c_fc_w_0, h_mlp_c_fc_w_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_mlp_c_fc_b_0.data(), d_mlp_c_fc_b_0, h_mlp_c_fc_b_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_mlp_c_proj_w_0.data(), d_mlp_c_proj_w_0, h_mlp_c_proj_w_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_mlp_c_proj_b_0.data(), d_mlp_c_proj_b_0, h_mlp_c_proj_b_0.size() * sizeof(float), hipMemcpyDeviceToHost));
}