#include "layer.cuh"

#include <hip/hip_runtime.h>

#include <stdexcept>

#include "utils.cuh"
#include "io.cuh"
#include "kernels.cuh"

Layer::Layer(uint64_t n_embd, uint64_t n_head) : 
        n_embd(n_embd),
        n_head(n_head),
        d_attn_c_attn_w_0(nullptr),
        d_attn_c_attn_b_0(nullptr),
        d_attn_c_proj_w_0(nullptr),
        d_attn_c_proj_b_0(nullptr),
        d_ln_1_b_0(nullptr),
        d_ln_1_g_0(nullptr),
        d_ln_2_b_0(nullptr),
        d_ln_2_g_0(nullptr),
        d_mlp_c_fc_w_0(nullptr),
        d_mlp_c_fc_b_0(nullptr),
        d_mlp_c_proj_w_0(nullptr),
        d_mlp_c_proj_b_0(nullptr) {
    // Allocate memory on host
    h_attn_c_attn_w_0.resize(n_embd * 3 * n_embd);
    h_attn_c_attn_b_0.resize(3 * n_embd);
    h_attn_c_proj_w_0.resize(n_embd * n_embd);
    h_attn_c_proj_b_0.resize(n_embd);
    h_ln_1_b_0.resize(n_embd);
    h_ln_1_g_0.resize(n_embd);
    h_ln_2_b_0.resize(n_embd);
    h_ln_2_g_0.resize(n_embd);
    h_mlp_c_fc_w_0.resize(n_embd * 4 * n_embd);
    h_mlp_c_fc_b_0.resize(4 * n_embd);
    h_mlp_c_proj_w_0.resize(4 * n_embd * n_embd);
    h_mlp_c_proj_b_0.resize(n_embd);

    // Allocate memory on device
    CHECK_CUDA(hipMalloc(&d_attn_c_attn_w_0, n_embd * 3 * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_attn_c_attn_b_0, 3 * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_attn_c_proj_w_0, n_embd * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_attn_c_proj_b_0, n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_ln_1_b_0, n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_ln_1_g_0, n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_ln_2_b_0, n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_ln_2_g_0, n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_fc_w_0, n_embd * 4 * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_fc_b_0, 4 * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_proj_w_0, 4 * n_embd * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_proj_b_0, n_embd * sizeof(fp_t)));
}

Layer::~Layer() {
    // Free memory on device
    CHECK_CUDA(hipFree(d_attn_c_attn_w_0));
    CHECK_CUDA(hipFree(d_attn_c_attn_b_0));
    CHECK_CUDA(hipFree(d_attn_c_proj_w_0));
    CHECK_CUDA(hipFree(d_attn_c_proj_b_0));
    CHECK_CUDA(hipFree(d_ln_1_b_0));
    CHECK_CUDA(hipFree(d_ln_1_g_0));
    CHECK_CUDA(hipFree(d_ln_2_b_0));
    CHECK_CUDA(hipFree(d_ln_2_g_0));
    CHECK_CUDA(hipFree(d_mlp_c_fc_w_0));
    CHECK_CUDA(hipFree(d_mlp_c_fc_b_0));
    CHECK_CUDA(hipFree(d_mlp_c_proj_w_0));
    CHECK_CUDA(hipFree(d_mlp_c_proj_b_0));
}

void Layer::apply(
        fp_t* d_hidden_states,
        fp_t* d_residual,
        fp_t* d_temp,
        uint64_t batch_size,
        uint64_t seq_length) {
    // Dimensions
    dim3 grid_size;
    dim3 block_size;

    // Allocate temporary buffers
    fp_t* d_qkv = nullptr;
    CHECK_CUDA(hipMalloc(&d_qkv, batch_size * seq_length * 3 * n_embd * sizeof(fp_t)));
    
    // Step 1: Save input for residual connection
    CHECK_CUDA(hipMemcpy(d_residual, d_hidden_states, batch_size * seq_length * n_embd * sizeof(fp_t), hipMemcpyDeviceToDevice));
    
    // Step 2: First layer normalization
    // Each thread handles one token (i_batch, i_sequence, :)
    // in the hidden states (batch, sequence, embedding)
    block_size.x = 32;
    block_size.y = 32;
    block_size.z = 1;
    grid_size.x = (batch_size + block_size.x - 1) / block_size.x;
    grid_size.y = (seq_length + block_size.y - 1) / block_size.y;
    grid_size.z = 1;
    layer_normalization_kernel<<<grid_size, block_size>>>(
        d_hidden_states, d_ln_1_g_0, d_ln_1_b_0,
        batch_size, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());

    // Step 3: Multi-head attention
    // Step 3.1: QKV projection
    // Each thread handles one token (i_batch, i_sequence, :)
    // in the QKV (batch, sequence, embedding)
    block_size.x = 32;
    block_size.y = 32;
    block_size.z = 1;
    grid_size.x = (batch_size + block_size.x - 1) / block_size.x;
    grid_size.y = (seq_length + block_size.y - 1) / block_size.y;
    grid_size.z = 1;
    qkv_projection_kernel<<<grid_size, block_size>>>(
        d_hidden_states, d_qkv, 
        d_attn_c_attn_w_0, d_attn_c_attn_b_0, 
        batch_size, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());

    // Step 3.2: Multi-head attention
    // Each thread handles one token (i_batch, i_sequence, :)
    // in the hidden states (batch, sequence, embedding)
    block_size.x = 32;
    block_size.y = 32;
    block_size.z = 1;
    grid_size.x = (batch_size + block_size.x - 1) / block_size.x;
    grid_size.y = (seq_length + block_size.y - 1) / block_size.y;
    grid_size.z = 1;
    multi_head_attention_kernel<<<grid_size, block_size>>>(
        d_qkv, d_hidden_states, 
        batch_size, seq_length, n_head, n_embd);
    CHECK_CUDA(hipGetLastError());

    // Step 3.3: Final projection
    // Each thread handles one token (i_batch, i_sequence, :)
    // in the hidden states (batch, sequence, embedding)
    block_size.x = 32;
    block_size.y = 32;
    block_size.z = 1;
    grid_size.x = (batch_size + block_size.x - 1) / block_size.x;
    grid_size.y = (seq_length + block_size.y - 1) / block_size.y;
    grid_size.z = 1;
    final_projection_kernel<<<grid_size, block_size>>>(
        d_hidden_states, d_temp,
        d_attn_c_proj_w_0, d_attn_c_proj_b_0,
        batch_size, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());

    // Step 4: Add residual connection
    // Each thread handles one token (i_batch, i_sequence, :)
    // in the hidden states (batch, sequence, embedding)
    block_size.x = 32;
    block_size.y = 32;
    block_size.z = 1;
    grid_size.x = (batch_size + block_size.x - 1) / block_size.x;
    grid_size.y = (seq_length + block_size.y - 1) / block_size.y;
    grid_size.z = 1;
    add_residual_kernel<<<grid_size, block_size>>>(
        d_temp, d_residual, d_hidden_states,
        batch_size, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());

    // Step 5: Save output for residual connection
    CHECK_CUDA(hipMemcpy(d_residual, d_hidden_states, seq_length * n_embd * sizeof(fp_t), hipMemcpyDeviceToDevice));

    // Step 6: Second layer normalization
    // Each thread handles one token (i_batch, i_sequence, :)
    // in the hidden states (batch, sequence, embedding)
    block_size.x = 32;
    block_size.y = 32;
    block_size.z = 1;
    grid_size.x = (batch_size + block_size.x - 1) / block_size.x;
    grid_size.y = (seq_length + block_size.y - 1) / block_size.y;
    grid_size.z = 1;
    layer_normalization_kernel<<<grid_size, block_size>>>(
        d_hidden_states, d_ln_2_g_0, d_ln_2_b_0,
        batch_size, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());

    // Step 7: MLP (feedforward network)
    // Each thread handles one token (i_batch, i_sequence, :)
    // in the hidden states (batch, sequence, embedding)
    block_size.x = 32;
    block_size.y = 32;
    block_size.z = 1;
    grid_size.x = (batch_size + block_size.x - 1) / block_size.x;
    grid_size.y = (seq_length + block_size.y - 1) / block_size.y;
    grid_size.z = 1;
    mlp_kernel<<<grid_size, block_size>>>(
        d_hidden_states, d_temp,
        d_mlp_c_fc_w_0, d_mlp_c_fc_b_0,
        d_mlp_c_proj_w_0, d_mlp_c_proj_b_0,
        batch_size, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());

    // Step 8: Add residual connection
    // Each thread handles one token (i_batch, i_sequence, :)
    // in the hidden states (batch, sequence, embedding)
    block_size.x = 32;
    block_size.y = 32;
    block_size.z = 1;
    grid_size.x = (batch_size + block_size.x - 1) / block_size.x;
    grid_size.y = (seq_length + block_size.y - 1) / block_size.y;
    grid_size.z = 1;
    add_residual_kernel<<<grid_size, block_size>>>(
        d_temp, d_residual, d_hidden_states,
        batch_size, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());

    // Free temporary buffers
    CHECK_CUDA(hipFree(d_qkv));
}

void Layer::load_from_hdf5(hid_t file_id, const std::string& layer_path) {
    read_dataset(file_id, layer_path + "/attn/c_attn/w_0", h_attn_c_attn_w_0);
    read_dataset(file_id, layer_path + "/attn/c_attn/b_0", h_attn_c_attn_b_0);
    read_dataset(file_id, layer_path + "/attn/c_proj/w_0", h_attn_c_proj_w_0);
    read_dataset(file_id, layer_path + "/attn/c_proj/b_0", h_attn_c_proj_b_0);
    read_dataset(file_id, layer_path + "/ln_1/b_0",        h_ln_1_b_0);
    read_dataset(file_id, layer_path + "/ln_1/g_0",        h_ln_1_g_0);
    read_dataset(file_id, layer_path + "/ln_2/b_0",        h_ln_2_b_0);
    read_dataset(file_id, layer_path + "/ln_2/g_0",        h_ln_2_g_0);
    read_dataset(file_id, layer_path + "/mlp/c_fc/w_0",    h_mlp_c_fc_w_0);
    read_dataset(file_id, layer_path + "/mlp/c_fc/b_0",    h_mlp_c_fc_b_0);
    read_dataset(file_id, layer_path + "/mlp/c_proj/w_0",  h_mlp_c_proj_w_0);
    read_dataset(file_id, layer_path + "/mlp/c_proj/b_0",  h_mlp_c_proj_b_0);
}

void Layer::copy_host_to_device() {
    CHECK_CUDA(hipMemcpy(d_attn_c_attn_w_0, h_attn_c_attn_w_0.data(), h_attn_c_attn_w_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_attn_b_0, h_attn_c_attn_b_0.data(), h_attn_c_attn_b_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_proj_w_0, h_attn_c_proj_w_0.data(), h_attn_c_proj_w_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_proj_b_0, h_attn_c_proj_b_0.data(), h_attn_c_proj_b_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_1_b_0, h_ln_1_b_0.data(), h_ln_1_b_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_1_g_0, h_ln_1_g_0.data(), h_ln_1_g_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_2_b_0, h_ln_2_b_0.data(), h_ln_2_b_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_2_g_0, h_ln_2_g_0.data(), h_ln_2_g_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_fc_w_0, h_mlp_c_fc_w_0.data(), h_mlp_c_fc_w_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_fc_b_0, h_mlp_c_fc_b_0.data(), h_mlp_c_fc_b_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_proj_w_0, h_mlp_c_proj_w_0.data(), h_mlp_c_proj_w_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_proj_b_0, h_mlp_c_proj_b_0.data(), h_mlp_c_proj_b_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
}

void Layer::copy_device_to_host() {
    CHECK_CUDA(hipMemcpy(h_attn_c_attn_w_0.data(), d_attn_c_attn_w_0, h_attn_c_attn_w_0.size() * sizeof(fp_t), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_attn_c_attn_b_0.data(), d_attn_c_attn_b_0, h_attn_c_attn_b_0.size() * sizeof(fp_t), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_attn_c_proj_w_0.data(), d_attn_c_proj_w_0, h_attn_c_proj_w_0.size() * sizeof(fp_t), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_attn_c_proj_b_0.data(), d_attn_c_proj_b_0, h_attn_c_proj_b_0.size() * sizeof(fp_t), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_ln_1_b_0.data(), d_ln_1_b_0, h_ln_1_b_0.size() * sizeof(fp_t), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_ln_1_g_0.data(), d_ln_1_g_0, h_ln_1_g_0.size() * sizeof(fp_t), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_ln_2_b_0.data(), d_ln_2_b_0, h_ln_2_b_0.size() * sizeof(fp_t), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_ln_2_g_0.data(), d_ln_2_g_0, h_ln_2_g_0.size() * sizeof(fp_t), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_mlp_c_fc_w_0.data(), d_mlp_c_fc_w_0, h_mlp_c_fc_w_0.size() * sizeof(fp_t), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_mlp_c_fc_b_0.data(), d_mlp_c_fc_b_0, h_mlp_c_fc_b_0.size() * sizeof(fp_t), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_mlp_c_proj_w_0.data(), d_mlp_c_proj_w_0, h_mlp_c_proj_w_0.size() * sizeof(fp_t), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_mlp_c_proj_b_0.data(), d_mlp_c_proj_b_0, h_mlp_c_proj_b_0.size() * sizeof(fp_t), hipMemcpyDeviceToHost));
}