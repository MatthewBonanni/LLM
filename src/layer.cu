#include "hip/hip_runtime.h"
#include "layer.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <stdexcept>
#include <vector>

#include "utils.cuh"
#include "io.cuh"
#include "kernels.cuh"

Layer::Layer(uint32_t n_ctx, uint32_t n_embd, uint32_t n_head) : 
        n_ctx(n_ctx),
        n_embd(n_embd),
        n_head(n_head),
        d_attn_c_attn_w_Q_0(nullptr),
        d_attn_c_attn_w_K_0(nullptr),
        d_attn_c_attn_w_V_0(nullptr),
        d_attn_c_attn_b_Q_0(nullptr),
        d_attn_c_attn_b_K_0(nullptr),
        d_attn_c_attn_b_V_0(nullptr),
        d_attn_c_proj_w_0(nullptr),
        d_attn_c_proj_b_0(nullptr),
        d_ln_1_b_0(nullptr),
        d_ln_1_g_0(nullptr),
        d_ln_2_b_0(nullptr),
        d_ln_2_g_0(nullptr),
        d_mlp_c_fc_w_0(nullptr),
        d_mlp_c_fc_b_0(nullptr),
        d_mlp_c_proj_w_0(nullptr),
        d_mlp_c_proj_b_0(nullptr),
        d_k_cache(nullptr),
        d_v_cache(nullptr),
        kv_cache_size(0) {
    // Allocate memory on device
    CHECK_CUDA(hipMalloc(&d_attn_c_attn_w_Q_0, n_embd * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_attn_c_attn_w_K_0, n_embd * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_attn_c_attn_w_V_0, n_embd * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_attn_c_attn_b_Q_0, n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_attn_c_attn_b_K_0, n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_attn_c_attn_b_V_0, n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_attn_c_proj_w_0, n_embd * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_attn_c_proj_b_0, n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_ln_1_b_0, n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_ln_1_g_0, n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_ln_2_b_0, n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_ln_2_g_0, n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_fc_w_0, n_embd * 4 * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_fc_b_0, 4 * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_proj_w_0, 4 * n_embd * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_proj_b_0, n_embd * sizeof(fp_t)));
}

Layer::~Layer() {
    // Free memory on device
    std::vector<void*> buffers = {
        d_attn_c_attn_w_Q_0,
        d_attn_c_attn_w_K_0,
        d_attn_c_attn_w_V_0,
        d_attn_c_attn_b_Q_0,
        d_attn_c_attn_b_K_0,
        d_attn_c_attn_b_V_0,
        d_attn_c_proj_w_0,
        d_attn_c_proj_b_0,
        d_ln_1_b_0,
        d_ln_1_g_0,
        d_ln_2_b_0,
        d_ln_2_g_0,
        d_mlp_c_fc_w_0,
        d_mlp_c_fc_b_0,
        d_mlp_c_proj_w_0,
        d_mlp_c_proj_b_0,
        d_k_cache,
        d_v_cache
    };
    clean_up_memory(buffers);
}

void Layer::allocate_kv_cache(uint32_t batch_size) {
    CHECK_CUDA(hipMalloc(&d_k_cache, (uint64_t)batch_size * n_ctx * n_embd * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_v_cache, (uint64_t)batch_size * n_ctx * n_embd * sizeof(half)));
}

void Layer::launch_layer_normalization(
        fp_t* d_input,
        uint32_t batch_size,
        uint32_t seq_length) {
    // Each block handles one token (i_batch, i_sequence, :)
    // in the hidden states (batch, sequence, embedding)
    dim3 grid_size(batch_size, seq_length, 1);
    dim3 block_size(256, 1, 1);
    layer_normalization_kernel<256, 8><<<grid_size, block_size>>>(
        d_input, d_ln_1_g_0, d_ln_1_b_0,
        batch_size, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());
}

void Layer::launch_qkv_projection(
        const fp_t* d_hidden_states,
        fp_t* d_q,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t seq_offset) {
    dim3 block_size_q(32, 32, 1);
    dim3 grid_size_q((n_embd     + block_size_q.x - 1) / block_size_q.x,
                     (batch_size + block_size_q.y - 1) / block_size_q.y,
                     1);
    q_projection_kernel<<<grid_size_q, block_size_q>>>(
        d_hidden_states, d_q,
        d_attn_c_attn_w_Q_0, d_attn_c_attn_b_Q_0,
        batch_size, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());

    // // DEBUG
    // std::vector<fp_t> h_q(batch_size * n_embd);
    // CHECK_CUDA(hipMemcpy(h_q.data(), d_q, batch_size * n_embd * sizeof(fp_t), hipMemcpyDeviceToHost));
    // std::cout << "Q: " << std::endl;
    // for (uint32_t i = 0; i < batch_size; i++) {
    //     for (uint32_t j = 0; j < 5; j++) {
    //         std::cout << h_q[i * n_embd + j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << "------------------" << std::endl;


    // constexpr uint32_t BLOCK_M_Q = 32;
    // constexpr uint32_t BLOCK_N_Q = 32;
    // constexpr uint32_t BLOCK_K_Q = 32;
    // constexpr uint32_t WARPS_PER_BLOCK_Q = (BLOCK_M_Q/WMMA_M * BLOCK_N_Q/WMMA_N);
    // dim3 grid_size_q(((n_embd     + BLOCK_N_Q - 1) / BLOCK_N_Q),
    //                  ((batch_size + BLOCK_M_Q - 1) / BLOCK_M_Q),
    //                  1);
    // dim3 block_size_q(WARP_SIZE, WARPS_PER_BLOCK_Q, 1);
    // uint32_t shared_mem_size_q = (BLOCK_M_Q * BLOCK_K_Q + BLOCK_K_Q * BLOCK_N_Q) * sizeof(half);
    // q_projection_kernel<BLOCK_M_Q, BLOCK_N_Q, BLOCK_K_Q><<<grid_size_q, block_size_q, shared_mem_size_q>>>(
    //     d_hidden_states, d_q,
    //     d_attn_c_attn_w_Q_0, d_attn_c_attn_b_Q_0,
    //     batch_size, seq_length, n_embd);
    // CHECK_CUDA(hipGetLastError());

    dim3 block_size_kv(32, 32, 1);
    dim3 grid_size_kv((n_embd     + block_size_kv.x - 1) / block_size_kv.x,
                      (seq_length + block_size_kv.y - 1) / block_size_kv.y,
                      batch_size);
    kv_projection_kernel<<<grid_size_kv, block_size_kv>>>(
        d_hidden_states, d_k_cache,
        d_attn_c_attn_w_K_0, d_attn_c_attn_b_K_0,
        batch_size, seq_length, seq_offset, n_embd);
    CHECK_CUDA(hipGetLastError());
    kv_projection_kernel<<<grid_size_kv, block_size_kv>>>(
        d_hidden_states, d_v_cache,
        d_attn_c_attn_w_V_0, d_attn_c_attn_b_V_0,
        batch_size, seq_length, seq_offset, n_embd);
    CHECK_CUDA(hipGetLastError());

    // // DEBUG
    // std::vector<half> h_k(batch_size * seq_length * n_embd);
    // CHECK_CUDA(hipMemcpy(h_k.data(), d_k_cache, batch_size * seq_length * n_embd * sizeof(half), hipMemcpyDeviceToHost));
    // std::cout << "seq_length: " << seq_length << std::endl;
    // std::cout << "seq_offset: " << seq_offset << std::endl;
    // std::cout << "K cache: " << std::endl;
    // for (uint32_t i = 0; i < batch_size; i++) {
    //     for (uint32_t j = 0; j < seq_length; j++) {
    //         for (uint32_t k = 0; k < 5; k++) {
    //             std::cout << __half2float(h_k[i * seq_length * n_embd + (j + seq_offset) * n_embd + k]) << " ";
    //         }
    //         std::cout << std::endl;
    //     }
    //     std::cout << "------------------" << std::endl;
    // }
    // exit(0);


    // constexpr uint32_t BLOCK_M_KV = 32;
    // constexpr uint32_t BLOCK_N_KV = 32;
    // constexpr uint32_t BLOCK_K_KV = 32;
    // constexpr uint32_t WARPS_PER_BLOCK_KV = (BLOCK_M_KV/WMMA_M * BLOCK_N_KV/WMMA_N);
    // dim3 grid_size_kv(((n_embd     + BLOCK_N_KV - 1) / BLOCK_N_KV),
    //                   ((seq_length + BLOCK_M_KV - 1) / BLOCK_M_KV),
    //                   batch_size);
    // dim3 block_size_kv(WARP_SIZE, WARPS_PER_BLOCK_KV, 1);
    // uint32_t shared_mem_size_kv = std::max(
    //     (BLOCK_M_KV * BLOCK_K_KV + BLOCK_K_KV * BLOCK_N_KV) * sizeof(half),
    //     (BLOCK_M_KV * BLOCK_N_KV) * sizeof(fp_t));
    // kv_projection_kernel<BLOCK_M_KV, BLOCK_N_KV, BLOCK_K_KV><<<grid_size_kv, block_size_kv, shared_mem_size_kv>>>(
    //     d_hidden_states, d_k_cache,
    //     d_attn_c_attn_w_K_0, d_attn_c_attn_b_K_0,
    //     batch_size, seq_length, seq_offset, n_embd);
    // CHECK_CUDA(hipGetLastError());
    // kv_projection_kernel<BLOCK_M_KV, BLOCK_N_KV, BLOCK_K_KV><<<grid_size_kv, block_size_kv, shared_mem_size_kv>>>(
    //     d_hidden_states, d_v_cache,
    //     d_attn_c_attn_w_V_0, d_attn_c_attn_b_V_0,
    //     batch_size, seq_length, seq_offset, n_embd);
    // CHECK_CUDA(hipGetLastError());
}

void Layer::launch_multi_head_attention(
        const fp_t* d_q,
        fp_t* d_output,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t seq_offset) {
    // Each thread handles one token (i_batch, i_sequence, :)
    // in the hidden states (batch, sequence, embedding)
    dim3 block_size(32, 32, 1);
    dim3 grid_size((batch_size + block_size.x - 1) / block_size.x,
                   (seq_length + block_size.y - 1) / block_size.y,
                   1);
    multi_head_attention_kernel<<<grid_size, block_size>>>(
        d_q, d_k_cache, d_v_cache, d_output,
        batch_size, seq_length, seq_offset, n_head, n_embd);
    CHECK_CUDA(hipGetLastError());
}

void Layer::launch_final_projection(
        const fp_t* d_input,
        fp_t* d_output,
        uint32_t batch_size,
        uint32_t seq_length) {
    // Each thread handles one token (i_batch, i_sequence, :)
    // in the hidden states (batch, sequence, embedding)
    dim3 block_size(32, 32, 1);
    dim3 grid_size((batch_size + block_size.x - 1) / block_size.x,
                   (seq_length + block_size.y - 1) / block_size.y,
                   1);
    final_projection_kernel<<<grid_size, block_size>>>(
        d_input, d_output,
        d_attn_c_proj_w_0, d_attn_c_proj_b_0,
        batch_size, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());
}

void Layer::launch_add_residual(
        const fp_t* d_input,
        const fp_t* d_residual,
        fp_t* d_output,
        uint32_t batch_size,
        uint32_t seq_length) {
    // Each block handles one token (i_batch, i_sequence, :)
    // in the hidden states (batch, sequence, embedding)
    dim3 block_size(128, 1, 1);
    dim3 grid_size(batch_size, seq_length, 1);
    add_residual_kernel<<<grid_size, block_size>>>(
        d_input, d_residual, d_output,
        batch_size, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());
}

void Layer::launch_mlp(
        const fp_t* d_input,
        fp_t* d_output,
        uint32_t batch_size,
        uint32_t seq_length) {
    // Each thread handles one token (i_batch, i_sequence, :)
    // in the hidden states (batch, sequence, embedding)
    dim3 block_size(32, 32, 1);
    dim3 grid_size((batch_size + block_size.x - 1) / block_size.x,
                   (seq_length + block_size.y - 1) / block_size.y,
                   1);
    mlp_kernel<<<grid_size, block_size>>>(
        d_input, d_output,
        d_mlp_c_fc_w_0, d_mlp_c_fc_b_0,
        d_mlp_c_proj_w_0, d_mlp_c_proj_b_0,
        batch_size, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());
}

void Layer::apply(
        fp_t* d_hidden_states,
        fp_t* d_residual,
        fp_t* d_temp,
        uint32_t batch_size,
        uint32_t seq_length,
        uint32_t seq_offset) {
    // Allocate temporary Q buffer
    fp_t* d_q = nullptr;
    CHECK_CUDA(hipMalloc(&d_q, (uint64_t)batch_size * n_embd * sizeof(fp_t)));
    
    // Step 1: Save input for residual connection
    CHECK_CUDA(hipMemcpy(
        d_residual,
        d_hidden_states,
        (uint64_t)batch_size * seq_length * n_embd * sizeof(fp_t),
        hipMemcpyDeviceToDevice));
    
    // Step 2: First layer normalization
    launch_layer_normalization(d_hidden_states, batch_size, seq_length);

    // Step 3: Multi-head attention
    // Step 3.1: QKV projection
    launch_qkv_projection(d_hidden_states, d_q, batch_size, seq_length, seq_offset);

    // Step 3.2: Multi-head attention
    launch_multi_head_attention(d_q, d_temp, batch_size, seq_length, seq_offset);

    // Step 3.3: Final projection
    launch_final_projection(d_temp, d_hidden_states, batch_size, seq_length);

    // Step 4: Add residual connection
    launch_add_residual(d_temp, d_residual, d_hidden_states, batch_size, seq_length);

    // Step 5: Save output for residual connection
    CHECK_CUDA(hipMemcpy(
        d_residual,
        d_hidden_states,
        (uint64_t)seq_length * n_embd * sizeof(fp_t),
        hipMemcpyDeviceToDevice));

    // Step 6: Second layer normalization
    launch_layer_normalization(d_hidden_states, batch_size, seq_length);

    // Step 7: MLP (feedforward network)
    launch_mlp(d_hidden_states, d_temp, batch_size, seq_length);

    // Step 8: Add residual connection
    launch_add_residual(d_temp, d_residual, d_hidden_states, batch_size, seq_length);

    // Free temporary buffer
    std::vector<void*> buffers = {d_q};
    clean_up_memory(buffers);
}

void Layer::load_from_hdf5(hid_t file_id, const std::string& layer_path) {
    // Host buffers
    std::vector<fp_t> h_attn_c_attn_w_0(n_embd * 3 * n_embd);
    std::vector<fp_t> h_attn_c_attn_b_0(3 * n_embd);
    std::vector<fp_t> h_attn_c_proj_w_0(n_embd * n_embd);
    std::vector<fp_t> h_attn_c_proj_b_0(n_embd);
    std::vector<fp_t> h_ln_1_b_0(n_embd);
    std::vector<fp_t> h_ln_1_g_0(n_embd);
    std::vector<fp_t> h_ln_2_b_0(n_embd);
    std::vector<fp_t> h_ln_2_g_0(n_embd);
    std::vector<fp_t> h_mlp_c_fc_w_0(n_embd * 4 * n_embd);
    std::vector<fp_t> h_mlp_c_fc_b_0(4 * n_embd);
    std::vector<fp_t> h_mlp_c_proj_w_0(4 * n_embd * n_embd);
    std::vector<fp_t> h_mlp_c_proj_b_0(n_embd);

    // Read datasets from HDF5 file
    read_dataset(file_id, layer_path + "/attn/c_attn/w_0", h_attn_c_attn_w_0);
    read_dataset(file_id, layer_path + "/attn/c_attn/b_0", h_attn_c_attn_b_0);
    read_dataset(file_id, layer_path + "/attn/c_proj/w_0", h_attn_c_proj_w_0);
    read_dataset(file_id, layer_path + "/attn/c_proj/b_0", h_attn_c_proj_b_0);
    read_dataset(file_id, layer_path + "/ln_1/b_0",        h_ln_1_b_0);
    read_dataset(file_id, layer_path + "/ln_1/g_0",        h_ln_1_g_0);
    read_dataset(file_id, layer_path + "/ln_2/b_0",        h_ln_2_b_0);
    read_dataset(file_id, layer_path + "/ln_2/g_0",        h_ln_2_g_0);
    read_dataset(file_id, layer_path + "/mlp/c_fc/w_0",    h_mlp_c_fc_w_0);
    read_dataset(file_id, layer_path + "/mlp/c_fc/b_0",    h_mlp_c_fc_b_0);
    read_dataset(file_id, layer_path + "/mlp/c_proj/w_0",  h_mlp_c_proj_w_0);
    read_dataset(file_id, layer_path + "/mlp/c_proj/b_0",  h_mlp_c_proj_b_0);

    // Separate attention weights and biases into Q, K, and V
    std::vector<fp_t> h_attn_c_attn_w_Q_0(n_embd * n_embd);
    std::vector<fp_t> h_attn_c_attn_w_K_0(n_embd * n_embd);
    std::vector<fp_t> h_attn_c_attn_w_V_0(n_embd * n_embd);
    std::vector<fp_t> h_attn_c_attn_b_Q_0(n_embd);
    std::vector<fp_t> h_attn_c_attn_b_K_0(n_embd);
    std::vector<fp_t> h_attn_c_attn_b_V_0(n_embd);
    for (uint32_t i = 0; i < n_embd; i++) {
        for (uint32_t j = 0; j < n_embd; j++) {
            // Store in column-major order
            h_attn_c_attn_w_Q_0[j * n_embd + i] = h_attn_c_attn_w_0[i * n_embd * 3 +              j];
            h_attn_c_attn_w_K_0[j * n_embd + i] = h_attn_c_attn_w_0[i * n_embd * 3 +     n_embd + j];
            h_attn_c_attn_w_V_0[j * n_embd + i] = h_attn_c_attn_w_0[i * n_embd * 3 + 2 * n_embd + j];
        }
        h_attn_c_attn_b_Q_0[i] = h_attn_c_attn_b_0[             i];
        h_attn_c_attn_b_K_0[i] = h_attn_c_attn_b_0[    n_embd + i];
        h_attn_c_attn_b_V_0[i] = h_attn_c_attn_b_0[2 * n_embd + i];
    }

    // Copy to device
    CHECK_CUDA(hipMemcpy(d_attn_c_attn_w_Q_0, h_attn_c_attn_w_Q_0.data(), h_attn_c_attn_w_Q_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_attn_w_K_0, h_attn_c_attn_w_K_0.data(), h_attn_c_attn_w_K_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_attn_w_V_0, h_attn_c_attn_w_V_0.data(), h_attn_c_attn_w_V_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_attn_b_Q_0, h_attn_c_attn_b_Q_0.data(), h_attn_c_attn_b_Q_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_attn_b_K_0, h_attn_c_attn_b_K_0.data(), h_attn_c_attn_b_K_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_attn_b_V_0, h_attn_c_attn_b_V_0.data(), h_attn_c_attn_b_V_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_proj_w_0, h_attn_c_proj_w_0.data(), h_attn_c_proj_w_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_proj_b_0, h_attn_c_proj_b_0.data(), h_attn_c_proj_b_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_1_b_0, h_ln_1_b_0.data(), h_ln_1_b_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_1_g_0, h_ln_1_g_0.data(), h_ln_1_g_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_2_b_0, h_ln_2_b_0.data(), h_ln_2_b_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_2_g_0, h_ln_2_g_0.data(), h_ln_2_g_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_fc_w_0, h_mlp_c_fc_w_0.data(), h_mlp_c_fc_w_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_fc_b_0, h_mlp_c_fc_b_0.data(), h_mlp_c_fc_b_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_proj_w_0, h_mlp_c_proj_w_0.data(), h_mlp_c_proj_w_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_proj_b_0, h_mlp_c_proj_b_0.data(), h_mlp_c_proj_b_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
}
