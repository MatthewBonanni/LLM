#include "hip/hip_runtime.h"
#include "layer.cuh"

#include <hip/hip_runtime.h>

#include <stdexcept>

#include "utils.cuh"
#include "io.cuh"

Layer::Layer(int n_embd, int n_head) : 
        n_embd(n_embd),
        n_head(n_head),
        d_attn_c_attn_w_0(nullptr),
        d_attn_c_attn_b_0(nullptr),
        d_attn_c_proj_w_0(nullptr),
        d_attn_c_proj_b_0(nullptr),
        d_ln_1_b_0(nullptr),
        d_ln_1_g_0(nullptr),
        d_ln_2_b_0(nullptr),
        d_ln_2_g_0(nullptr),
        d_mlp_c_fc_w_0(nullptr),
        d_mlp_c_fc_b_0(nullptr),
        d_mlp_c_proj_w_0(nullptr),
        d_mlp_c_proj_b_0(nullptr) {
    // Allocate memory on host
    h_attn_c_attn_w_0.resize(n_embd * 3 * n_embd);
    h_attn_c_attn_b_0.resize(3 * n_embd);
    h_attn_c_proj_w_0.resize(n_embd * n_embd);
    h_attn_c_proj_b_0.resize(n_embd);
    h_ln_1_b_0.resize(n_embd);
    h_ln_1_g_0.resize(n_embd);
    h_ln_2_b_0.resize(n_embd);
    h_ln_2_g_0.resize(n_embd);
    h_mlp_c_fc_w_0.resize(n_embd * 4 * n_embd);
    h_mlp_c_fc_b_0.resize(4 * n_embd);
    h_mlp_c_proj_w_0.resize(4 * n_embd * n_embd);
    h_mlp_c_proj_b_0.resize(n_embd);

    // Allocate memory on device
    CHECK_CUDA(hipMalloc(&d_attn_c_attn_w_0, n_embd * 3 * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_attn_c_attn_b_0, 3 * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_attn_c_proj_w_0, n_embd * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_attn_c_proj_b_0, n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_ln_1_b_0, n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_ln_1_g_0, n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_ln_2_b_0, n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_ln_2_g_0, n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_fc_w_0, n_embd * 4 * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_fc_b_0, 4 * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_proj_w_0, 4 * n_embd * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_mlp_c_proj_b_0, n_embd * sizeof(float)));
}

Layer::~Layer() {
    // Free memory on device
    CHECK_CUDA(hipFree(d_attn_c_attn_w_0));
    CHECK_CUDA(hipFree(d_attn_c_attn_b_0));
    CHECK_CUDA(hipFree(d_attn_c_proj_w_0));
    CHECK_CUDA(hipFree(d_attn_c_proj_b_0));
    CHECK_CUDA(hipFree(d_ln_1_b_0));
    CHECK_CUDA(hipFree(d_ln_1_g_0));
    CHECK_CUDA(hipFree(d_ln_2_b_0));
    CHECK_CUDA(hipFree(d_ln_2_g_0));
    CHECK_CUDA(hipFree(d_mlp_c_fc_w_0));
    CHECK_CUDA(hipFree(d_mlp_c_fc_b_0));
    CHECK_CUDA(hipFree(d_mlp_c_proj_w_0));
    CHECK_CUDA(hipFree(d_mlp_c_proj_b_0));
}

__device__ __host__ float gelu(float x) {
    return 0.5f * x * (1.0f + tanhf(0.7978845608f * (x + 0.044715f * x * x * x)));
}

__global__ void multi_head_attention(float* input, float* output, float* w_qkv, float* b_qkv, 
                                     float* w_proj, float* b_proj, int seq_length, int n_embd, int n_head) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i_token = idx; // Each thread processes one token
    if (i_token >= seq_length) {
        return;
    }
    
    // Calculate dimensions
    int d_k = n_embd / n_head; // Dimension of each head
    int qkv_size = 3 * n_embd; // Size of Q, K, V for each token
    
    // Allocate shared memory dynamically
    extern __shared__ float shared_mem[];
    float* qkv       = shared_mem;                             // Size: seq_length * qkv_size
    float* out_heads = qkv        + seq_length * qkv_size;     // Size: n_head * seq_length * d_k
    float* scores    = out_heads  + n_head * seq_length * d_k; // Size: seq_length
    
    // Linear projection to get Q, K, V for this token
    for (int i = 0; i < qkv_size; i++) {
        float val = b_qkv[i];
        for (int j = 0; j < n_embd; j++) {
            val += input[i_token * n_embd + j] * w_qkv[j * qkv_size + i];
        }
        qkv[i_token * qkv_size + i] = val;
    }
    
    __syncthreads();
    
    // Set up pointers to the Q, K, V sections in shared memory
    float* q_start = qkv;                           // Q starts at the beginning
    float* k_start = q_start + seq_length * n_embd; // K starts after all Q values
    float* v_start = k_start + seq_length * n_embd; // V starts after all K values
    
    float scale = 1.0f / sqrtf(d_k);
    
    // Process each attention head
    for (int i_head = 0; i_head < n_head; i_head++) {
        // Output pointer for this head and token
        float* out_ptr = out_heads + i_head * seq_length * d_k + i_token * d_k;
        
        // For current token (i_token) and current head (i_head)
        float* q_ptr = q_start + i_token * n_embd + i_head * d_k;
        
        // Calculate attention scores between current token and all other tokens
        float max_val = -INFINITY;
        for (int j_token = 0; j_token < seq_length; j_token++) {
            // Get K values for token j_token, head i_head
            float* k_ptr = k_start + j_token * n_embd + i_head * d_k;
            
            // Compute dot product
            float dot = 0.0f;
            for (int d = 0; d < d_k; d++) {
                dot += q_ptr[d] * k_ptr[d];
            }
            scores[j_token] = dot * scale;
            max_val = fmaxf(max_val, scores[j_token]);
        }
        
        // Softmax calculation for attention weights
        float sum = 0.0f;
        for (int j_token = 0; j_token < seq_length; j_token++) {
            scores[j_token] = expf(scores[j_token] - max_val);
            sum += scores[j_token];
        }
        
        for (int j_token = 0; j_token < seq_length; j_token++) {
            scores[j_token] /= sum;
        }
        
        // Calculate weighted sum of values
        for (int d = 0; d < d_k; d++) {
            float weighted_sum = 0.0f;
            for (int j_token = 0; j_token < seq_length; j_token++) {
                // Get V values for token j_token, head i_head
                float* v_ptr = v_start + j_token * n_embd + i_head * d_k;
                weighted_sum += scores[j_token] * v_ptr[d];
            }
            out_ptr[d] = weighted_sum;
        }
    }
    
    __syncthreads();
    
    // Linear projection to get final output
    for (int i = 0; i < n_embd; i++) {
        float val = b_proj[i];
        for (int i_head = 0; i_head < n_head; i_head++) {
            for (int d = 0; d < d_k; d++) {
                int j = i_head * d_k + d;  // Concatenated head outputs
                val += out_heads[i_head * seq_length * d_k + i_token * d_k + d] *
                       w_proj[i * n_embd + j];
            }
        }
        output[i_token * n_embd + i] = val;
    }
}

__global__ void layer_normalization(float* input, float* output, float* gamma, float* beta, int seq_length, int n_embd) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i_token = idx; // Each thread processes one token
    if (i_token >= seq_length) {
        return;
    }
    
    // Calculate mean
    float mean = 0.0f;
    for (int i = 0; i < n_embd; i++) {
        mean += input[i_token * n_embd + i];
    }
    mean /= n_embd;
    
    // Calculate variance
    float var = 0.0f;
    for (int i = 0; i < n_embd; i++) {
        float diff = input[i_token * n_embd + i] - mean;
        var += diff * diff;
    }
    var /= n_embd;
    
    // Normalize and scale
    const float epsilon = 1e-5f;
    float inv_std = 1.0f / sqrtf(var + epsilon);
    
    for (int i = 0; i < n_embd; i++) {
        float normalized = (input[i_token * n_embd + i] - mean) * inv_std;
        output[i_token * n_embd + i] = gamma[i] * normalized + beta[i];
    }
}

__global__ void mlp(float* input, float* output, float* w_fc, float* b_fc, 
                    float* w_proj, float* b_proj, int seq_length, int n_embd) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i_token = idx; // Each thread processes one token
    if (i_token >= seq_length) {
        return;
    }

    int intermediate_size = 4 * n_embd;
    float intermediate[4096]; // Assuming max intermediate_size = 4096, adjust as needed

    // Compute feedforward layer
    for (int i = 0; i < intermediate_size; i++) {
        float val = b_fc[i];
        for (int j = 0; j < n_embd; j++) {
            val += input[i_token * n_embd + j] * w_fc[j * intermediate_size + i];
        }
        intermediate[i] = gelu(val);
    }

    // Compute projection back to hidden size
    for (int i = 0; i < n_embd; i++) {
        float val = b_proj[i];
        for (int j = 0; j < intermediate_size; j++) {
            val += intermediate[j] * w_proj[j * n_embd + i];
        }
        output[i_token * n_embd + i] = val;
    }
}

__global__ void add_residual(float* input, float* residual, float* output, int seq_length, int n_embd) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i_token = idx; // Each thread processes one token
    if (i_token >= seq_length) {
        return;
    }
    
    for (int i = 0; i < n_embd; i++) {
        output[i_token * n_embd + i] = input[i_token * n_embd + i] + residual[i_token * n_embd + i];
    }
}

void Layer::apply(float* d_hidden_states, float* d_residual, float* d_temp, int seq_length) {
    // Calculate dimensions
    int block_size = 256; // Using a fixed block size that works well for most cases
    int grid_size = (seq_length * n_embd + block_size - 1) / block_size;
    int d_k = n_embd / n_head;
    
    // Calculate shared memory for multi-head attention
    int shared_mem_size = seq_length * 3 * n_embd +   // qkv storage
                          n_head * seq_length * d_k + // output heads storage
                          seq_length;                 // attention scores storage
    
    // Step 1: Save input for residual connection
    std::cout << "> STEP 1: Saving input for residual connection" << std::endl;
    CHECK_CUDA(hipMemcpy(d_residual, d_hidden_states, seq_length * n_embd * sizeof(float), hipMemcpyDeviceToDevice));
    
    // Step 2: First layer normalization
    std::cout << "> STEP 2: First layer normalization" << std::endl;
    layer_normalization<<<grid_size, block_size>>>(d_hidden_states, d_temp, d_ln_1_g_0, d_ln_1_b_0, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());
    
    // Step 3: Multi-head attention
    std::cout << "> STEP 3: Multi-head attention" << std::endl;
    multi_head_attention<<<grid_size, block_size, shared_mem_size * sizeof(float)>>>(
        d_temp, d_hidden_states, 
        d_attn_c_attn_w_0, d_attn_c_attn_b_0, 
        d_attn_c_proj_w_0, d_attn_c_proj_b_0, 
        seq_length, n_embd, n_head);
    CHECK_CUDA(hipGetLastError());
    
    // Step 4: Add residual connection
    std::cout << "> STEP 4: Add residual connection" << std::endl;
    add_residual<<<grid_size, block_size>>>(d_hidden_states, d_residual, d_hidden_states, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());
    
    // Step 5: Save output for residual connection
    std::cout << "> STEP 5: Saving output for residual connection" << std::endl;
    CHECK_CUDA(hipMemcpy(d_residual, d_hidden_states, seq_length * n_embd * sizeof(float), hipMemcpyDeviceToDevice));
    
    // Step 6: Second layer normalization
    std::cout << "> STEP 6: Second layer normalization" << std::endl;
    layer_normalization<<<grid_size, block_size>>>(d_hidden_states, d_temp, d_ln_2_g_0, d_ln_2_b_0, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());
    
    // Step 7: MLP (feedforward network)
    std::cout << "> STEP 7: MLP (feedforward network)" << std::endl;
    mlp<<<grid_size, block_size>>>(
        d_temp, d_hidden_states,
        d_mlp_c_fc_w_0, d_mlp_c_fc_b_0,
        d_mlp_c_proj_w_0, d_mlp_c_proj_b_0,
        seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());
    
    // Step 8: Add residual connection
    std::cout << "> STEP 8: Add residual connection" << std::endl;
    add_residual<<<grid_size, block_size>>>(d_hidden_states, d_residual, d_hidden_states, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());
    
    // Final synchronization (only if needed)
    // CHECK_CUDA(hipDeviceSynchronize());
}

void Layer::load_from_hdf5(hid_t file_id, const std::string& layer_path) {
    read_dataset(file_id, layer_path + "/attn/c_attn/w_0", h_attn_c_attn_w_0);
    read_dataset(file_id, layer_path + "/attn/c_attn/b_0", h_attn_c_attn_b_0);
    read_dataset(file_id, layer_path + "/attn/c_proj/w_0", h_attn_c_proj_w_0);
    read_dataset(file_id, layer_path + "/attn/c_proj/b_0", h_attn_c_proj_b_0);
    read_dataset(file_id, layer_path + "/ln_1/b_0",        h_ln_1_b_0);
    read_dataset(file_id, layer_path + "/ln_1/g_0",        h_ln_1_g_0);
    read_dataset(file_id, layer_path + "/ln_2/b_0",        h_ln_2_b_0);
    read_dataset(file_id, layer_path + "/ln_2/g_0",        h_ln_2_g_0);
    read_dataset(file_id, layer_path + "/mlp/c_fc/w_0",    h_mlp_c_fc_w_0);
    read_dataset(file_id, layer_path + "/mlp/c_fc/b_0",    h_mlp_c_fc_b_0);
    read_dataset(file_id, layer_path + "/mlp/c_proj/w_0",  h_mlp_c_proj_w_0);
    read_dataset(file_id, layer_path + "/mlp/c_proj/b_0",  h_mlp_c_proj_b_0);
}

void Layer::copy_host_to_device() {
    CHECK_CUDA(hipMemcpy(d_attn_c_attn_w_0, h_attn_c_attn_w_0.data(), h_attn_c_attn_w_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_attn_b_0, h_attn_c_attn_b_0.data(), h_attn_c_attn_b_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_proj_w_0, h_attn_c_proj_w_0.data(), h_attn_c_proj_w_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_attn_c_proj_b_0, h_attn_c_proj_b_0.data(), h_attn_c_proj_b_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_1_b_0, h_ln_1_b_0.data(), h_ln_1_b_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_1_g_0, h_ln_1_g_0.data(), h_ln_1_g_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_2_b_0, h_ln_2_b_0.data(), h_ln_2_b_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_2_g_0, h_ln_2_g_0.data(), h_ln_2_g_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_fc_w_0, h_mlp_c_fc_w_0.data(), h_mlp_c_fc_w_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_fc_b_0, h_mlp_c_fc_b_0.data(), h_mlp_c_fc_b_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_proj_w_0, h_mlp_c_proj_w_0.data(), h_mlp_c_proj_w_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mlp_c_proj_b_0, h_mlp_c_proj_b_0.data(), h_mlp_c_proj_b_0.size() * sizeof(float), hipMemcpyHostToDevice));
}

void Layer::copy_device_to_host() {
    CHECK_CUDA(hipMemcpy(h_attn_c_attn_w_0.data(), d_attn_c_attn_w_0, h_attn_c_attn_w_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_attn_c_attn_b_0.data(), d_attn_c_attn_b_0, h_attn_c_attn_b_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_attn_c_proj_w_0.data(), d_attn_c_proj_w_0, h_attn_c_proj_w_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_attn_c_proj_b_0.data(), d_attn_c_proj_b_0, h_attn_c_proj_b_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_ln_1_b_0.data(), d_ln_1_b_0, h_ln_1_b_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_ln_1_g_0.data(), d_ln_1_g_0, h_ln_1_g_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_ln_2_b_0.data(), d_ln_2_b_0, h_ln_2_b_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_ln_2_g_0.data(), d_ln_2_g_0, h_ln_2_g_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_mlp_c_fc_w_0.data(), d_mlp_c_fc_w_0, h_mlp_c_fc_w_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_mlp_c_fc_b_0.data(), d_mlp_c_fc_b_0, h_mlp_c_fc_b_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_mlp_c_proj_w_0.data(), d_mlp_c_proj_w_0, h_mlp_c_proj_w_0.size() * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_mlp_c_proj_b_0.data(), d_mlp_c_proj_b_0, h_mlp_c_proj_b_0.size() * sizeof(float), hipMemcpyDeviceToHost));
}