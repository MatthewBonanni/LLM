#include "hip/hip_runtime.h"
#include "llm.cuh"

#include <hip/hip_runtime.h>

#include <fstream>
#include <iostream>
#include <stdexcept>
#include <string>
#include <vector>

#include <nlohmann/json.hpp>
#include <hdf5_hl.h>

#include "utils.cuh"
#include "io.cuh"
#include "tokenizer.cuh"
#include "layer.cuh"

LLM::LLM(const std::string& model_path) :
        tokenizer(model_path),
        d_wte_0(nullptr),
        d_wpe_0(nullptr),
        d_ln_f_b_0(nullptr),
        d_ln_f_g_0(nullptr),
        max_out_length(50),
        temperature(0.8f),
        n_top_predictions(10) {
    load_hparams(model_path);
    load_model(model_path);
}

LLM::~LLM() {}

void LLM::print() {
    printf("--------------------------------\n");
    printf("LLM Configuration\n");
    printf("--------------------------------\n");
    printf("n_vocab: %d\n", n_vocab);
    printf("n_ctx: %d\n", n_ctx);
    printf("n_embd: %d\n", n_embd);
    printf("n_head: %d\n", n_head);
    printf("n_layer: %d\n", n_layer);
    printf("--------------------------------\n");
}

void LLM::load_hparams(std::string model_path) {
    std::ifstream file(model_path + "/hparams.json");
    if (!file) {
        std::cerr << "Error: Cannot open encoder.json" << std::endl;
        return;
    }

    nlohmann::json j;
    file >> j;
    n_vocab = j["n_vocab"];
    n_ctx = j["n_ctx"];
    n_embd = j["n_embd"];
    n_head = j["n_head"];
    n_layer = j["n_layer"];
}

void LLM::load_model(std::string model_path) {
    std::cout << "Loading model weights from " << model_path << std::endl;
    hid_t file_id = H5Fopen((model_path + "/model.h5").c_str(), H5F_ACC_RDONLY, H5P_DEFAULT);
    if (file_id < 0) {
        throw std::runtime_error("Error: Cannot open model.h5");
    }

    std::string base_path = "/model_weights/model";

    // Load embeddings
    std::cout << "> Embeddings" << std::endl;
    std::cout << "  > Allocating host and device memory..." << std::endl;
    h_wte_0.resize(n_vocab * n_embd);
    h_wpe_0.resize(n_ctx * n_embd);
    CHECK_CUDA(hipMalloc(&d_wte_0, n_vocab * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_wpe_0, n_ctx * n_embd * sizeof(float)));
    std::cout << "  > Loading weights..." << std::endl;
    read_dataset(file_id, base_path + "/wte_0", h_wte_0);
    read_dataset(file_id, base_path + "/wpe_0", h_wpe_0);

    // Load layers
    for (int i = 0; i < n_layer; i++) {
        std::cout << "> Layer " << i << std::endl;
        std::cout << "  > Allocating host and device memory..." << std::endl;
        layers.push_back(std::make_unique<Layer>(n_embd, n_head));

        std::string layer_path = base_path + "/h" + std::to_string(i);
        std::cout << "  > Loading weights..." << std::endl;
        layers[i]->load_from_hdf5(file_id, layer_path);
    }

    // Load final layer norm
    std::cout << "> Final layer norm" << std::endl;
    std::cout << "  > Allocating host and device memory..." << std::endl;
    h_ln_f_b_0.resize(n_embd);
    h_ln_f_g_0.resize(n_embd);
    CHECK_CUDA(hipMalloc(&d_ln_f_b_0, n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_ln_f_g_0, n_embd * sizeof(float)));
    std::cout << "  > Loading weights..." << std::endl;
    read_dataset(file_id, base_path + "/ln_f/b_0", h_ln_f_b_0);
    read_dataset(file_id, base_path + "/ln_f/g_0", h_ln_f_g_0);

    // Close the file
    H5Fclose(file_id);

    // Copy weights to device
    copy_params_host_to_device();
}

__global__ void embedding_kernel(const int* token_ids,
                                 const float* wte,
                                 const float* wpe,
                                 float* embeddings,
                                 int token_count,
                                 int embedding_dim) {
    // Calculate global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if this thread should process an element
    if (idx < token_count * embedding_dim) {
        // Calculate which token and which embedding dimension this thread is handling
        int token_idx = idx / embedding_dim;    // Which token
        int embd_idx = idx % embedding_dim;     // Which dimension in the embedding

        // Get the token ID for this position
        int token_id = token_ids[token_idx];

        // Calculate offset in embedding tables
        int token_offset = token_id * embedding_dim + embd_idx;
        int pos_offset = token_idx * embedding_dim + embd_idx;

        // Sum token embedding and positional embedding
        embeddings[idx] = wte[token_offset] + wpe[pos_offset];
    }
}

void LLM::apply_embeddings(int* d_token_ids, float* d_embeddings, int token_count) {
    // Kernel to compute final embeddings
    int threads = 256;
    int blocks = (token_count * n_embd + threads - 1) / threads;
    embedding_kernel<<<blocks, threads>>>(d_token_ids, d_wte_0, d_wpe_0, d_embeddings, token_count, n_embd);
}

__global__ void layer_norm_kernel(float* hidden_states, const float* gamma, const float* beta,
                                  int seq_length, int hidden_size) {
    extern __shared__ float shared_data[];
    float* shared_sum = shared_data;
    float* shared_sum_sq = shared_data + blockDim.x;

    int pos = blockIdx.x;
    int tid = threadIdx.x;
    float* pos_hidden = hidden_states + pos * hidden_size;

    // Initialize shared memory
    shared_sum[tid] = 0.0f;
    shared_sum_sq[tid] = 0.0f;

    // Calculate partial sums for mean and variance
    for (int i = tid; i < hidden_size; i += blockDim.x) {
        float val = pos_hidden[i];
        shared_sum[tid] += val;
        shared_sum_sq[tid] += val * val;
    }
    __syncthreads();

    // Parallel reduction for sum and sum of squares
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_sum[tid] += shared_sum[tid + stride];
            shared_sum_sq[tid] += shared_sum_sq[tid + stride];
        }
        __syncthreads();
    }

    // Calculate mean and variance
    const float epsilon = 1e-5f;
    float mean = shared_sum[0] / hidden_size;
    float var = (shared_sum_sq[0] / hidden_size) - (mean * mean) + epsilon;
    float inv_std = rsqrtf(var);

    // Apply normalization with gamma and beta
    for (int i = tid; i < hidden_size; i += blockDim.x) {
        float normalized = (pos_hidden[i] - mean) * inv_std;
        pos_hidden[i] = gamma[i] * normalized + beta[i];
    }
}

void LLM::apply_final_layer_norm(float* d_hidden_states, int seq_length) {
    // Launch one block per sequence position, with threads for hidden dimension
    dim3 grid(seq_length);
    dim3 block(256);
    size_t shared_mem_size = 2 * block.x * sizeof(float);

    layer_norm_kernel<<<grid, block, shared_mem_size>>>(
        d_hidden_states, d_ln_f_g_0, d_ln_f_b_0, seq_length, n_embd);
}

__global__ void lm_head_kernel(float* hidden_state, float* logits,
                               float* weights, float* biases,
                               int n_vocab, int n_embd) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_vocab) {
        logits[idx] = biases ? biases[idx] : 0.0f;
        for (int i = 0; i < n_embd; i++) {
            logits[idx] += hidden_state[i] * weights[idx * n_embd + i];
        }
    }
}

void LLM::apply_lm_head(float* d_hidden_state, float* d_logits) {
    // GPT-2 uses wte as the lm_head
    int threads = 256;
    int blocks = (n_vocab + threads - 1) / threads;
    lm_head_kernel<<<blocks, threads>>>(d_hidden_state, d_logits, d_wte_0, nullptr, n_vocab, n_embd);
}

void LLM::copy_params_host_to_device() {
    CHECK_CUDA(hipMemcpy(d_wte_0, h_wte_0.data(), h_wte_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_wpe_0, h_wpe_0.data(), h_wpe_0.size() * sizeof(float), hipMemcpyHostToDevice));
    for (int i = 0; i < n_layer; i++) {
        layers[i]->copy_host_to_device();
    }
}

void LLM::run_interactive() {
    std::cout << "LLM Running Mode. Use CTRL-C to quit.\n";

    while (true) {
        // Get user input
        std::string input;
        std::cout << ">> ";
        std::getline(std::cin, input);

        // Tokenize input
        std::vector<int> h_token_ids = tokenizer.tokenize(input);

        // Print token info
        std::cout << "Token IDs: ";
        for (int id : h_token_ids) {
            std::cout << id << " ";
        }
        std::cout << "\nToken count: " << h_token_ids.size() << std::endl;

        // If empty input, continue
        if (h_token_ids.empty()) {
            std::cout << "Empty input, please try again.\n";
            continue;
        }

        // If input is too long, truncate
        if (h_token_ids.size() > n_ctx) {
            h_token_ids.resize(n_ctx);
            std::cout << "Input too long, truncating to " << n_ctx << " tokens." << std::endl;
        }

        // Generate text
        generate_text(h_token_ids);
    }
}

std::vector<float> LLM::forward_pass(const std::vector<int>& tokens) {
    // Allocate device memory for token IDs and embeddings
    int* d_token_ids = nullptr;
    float* d_hidden_states = nullptr;
    float* d_residual = nullptr;
    float* d_temp = nullptr;
    float* d_logits = nullptr;
    std::vector<float> h_logits(n_vocab);

    std::cout << std::endl;
    std::cout << "Forward pass..." << std::endl;

    // Allocate device memory
    std::cout << "> Allocating device memory..." << std::endl;
    CHECK_CUDA(hipMalloc(&d_token_ids, tokens.size() * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_hidden_states, tokens.size() * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_residual, tokens.size() * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_temp, tokens.size() * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_logits, n_vocab * sizeof(float)));

    // Token IDs
    CHECK_CUDA(hipMemcpy(d_token_ids, tokens.data(), tokens.size() * sizeof(int), hipMemcpyHostToDevice));

    // Embeddings
    std::cout << "> Computing embeddings..." << std::endl;
    apply_embeddings(d_token_ids, d_hidden_states, tokens.size());
    d_token_ids = nullptr;

    // Process through transformer layers
    for (int i = 0; i < n_layer; i++) {
        std::cout << "> Applying layer " << i << "..." << std::endl;
        layers[i]->apply(d_hidden_states, d_residual, d_temp, tokens.size());
    }

    // Apply final layer norm
    std::cout << "> Applying final layer norm..." << std::endl;
    apply_final_layer_norm(d_hidden_states, tokens.size());

    // Get logits for the last token position
    std::cout << "> Applying LM head..." << std::endl;
    apply_lm_head(d_hidden_states + (tokens.size() - 1) * n_embd, d_logits);

    // Synchronize device
    CHECK_CUDA(hipDeviceSynchronize());

    // Copy logits to host
    CHECK_CUDA(hipMemcpy(h_logits.data(), d_logits, n_vocab * sizeof(float), hipMemcpyDeviceToHost));

    // Clean up resources
    clean_up_memory({d_token_ids, d_hidden_states, d_residual, d_logits});

    return h_logits;
}

std::vector<std::pair<float, int>> LLM::get_top_predictions(const std::vector<float>& logits) {
    std::vector<std::pair<float, int>> probs;
    probs.reserve(n_vocab);
    
    // Find max for numerical stability
    float max_logit = *std::max_element(logits.begin(), logits.end());
    float sum_exp = 0.0f;
    
    // Apply softmax
    for (int i = 0; i < n_vocab; i++) {
        float prob = std::exp(logits[i] - max_logit);
        sum_exp += prob;
        probs.push_back({prob, i});
    }
    
    // Normalize
    for (auto& p : probs) {
        p.first /= sum_exp;
    }
    
    // Sort by probability (descending)
    std::partial_sort(probs.begin(), probs.begin() + n_top_predictions, probs.end(),
                      [](const auto& a, const auto& b) { return a.first > b.first; });
    
    return probs;
}

int LLM::sample_token(const std::vector<std::pair<float, int>>& probs) {
    // If temperature is 0, do greedy sampling
    if (temperature == 0.0f) {
        return probs[0].second;
    }
    
    // Create a copy for temperature adjustment
    std::vector<std::pair<float, int>> temp_adjusted = probs;
    
    // Apply temperature adjustment
    if (temperature != 1.0f) {
        float sum = 0.0f;
        for (auto& p : temp_adjusted) {
            p.first = std::pow(p.first, 1.0f / temperature);
            sum += p.first;
        }
        // Renormalize
        for (auto& p : temp_adjusted) {
            p.first /= sum;
        }
    }
    
    // Sample based on adjusted probabilities
    // TODO - better random number generation
    float r = static_cast<float>(rand()) / RAND_MAX;
    float cdf = 0.0f;
    
    for (const auto& p : temp_adjusted) {
        cdf += p.first;
        if (r <= cdf) {
            return p.second;
        }
    }
    
    // Fallback to most likely token
    return temp_adjusted[0].second;
}

void LLM::generate_text(const std::vector<int>& input_ids) {
    std::cout << "Generated: ";
    std::vector<int> generated_tokens = input_ids;
    
    for (int gen_idx = 0; gen_idx < max_out_length; gen_idx++) {
        // Forward pass for the current sequence
        std::vector<float> logits = forward_pass(generated_tokens);
        
        // Get predictions
        auto predictions = get_top_predictions(logits);
        
        // Sample next token
        int next_token = sample_token(predictions);
        
        // Add to generated sequence
        generated_tokens.push_back(next_token);
        
        // Print the token
        std::string token_str = tokenizer.detokenize({next_token});
        std::cout << token_str;
        std::cout.flush();
        
        // Check for EOS token
        if (next_token == tokenizer.eos_token_id()) {
            break;
        }
    }
    
    std::cout << std::endl;
}

void LLM::clean_up_memory(const std::vector<void*>& buffers) {
    for (void* buffer : buffers) {
        if (buffer != nullptr) {
            CHECK_CUDA(hipFree(buffer));
        }
    }
}