#include "hip/hip_runtime.h"
#include "llm.cuh"

#include <hip/hip_runtime.h>

#include <fstream>
#include <iostream>
#include <stdexcept>
#include <string>
#include <vector>
#include <random>

#include <nlohmann/json.hpp>
#include <hdf5_hl.h>

#include "utils.cuh"
#include "io.cuh"
#include "tokenizer.cuh"
#include "layer.cuh"
#include "kernels.cuh"

LLM::LLM(const std::string& model_path) :
        tokenizer(model_path),
        d_wte_0(nullptr),
        d_wpe_0(nullptr),
        d_ln_f_b_0(nullptr),
        d_ln_f_g_0(nullptr),
        max_out_length(50),
        temperature(0.7f),
        n_top_predictions(200) {
    load_hparams(model_path);
    load_model(model_path);

    if (n_top_predictions > n_vocab) {
        n_top_predictions = n_vocab;
    }
}

LLM::~LLM() {}

void LLM::print() {
    printf("--------------------------------\n");
    printf("LLM Configuration\n");
    printf("--------------------------------\n");
    printf("n_vocab: %d\n", n_vocab);
    printf("n_ctx: %d\n", n_ctx);
    printf("n_embd: %d\n", n_embd);
    printf("n_head: %d\n", n_head);
    printf("n_layer: %d\n", n_layer);
    printf("--------------------------------\n");
}

void LLM::load_hparams(std::string model_path) {
    std::ifstream file(model_path + "/hparams.json");
    if (!file) {
        std::cerr << "Error: Cannot open encoder.json" << std::endl;
        return;
    }

    nlohmann::json j;
    file >> j;
    n_vocab = j["n_vocab"];
    n_ctx = j["n_ctx"];
    n_embd = j["n_embd"];
    n_head = j["n_head"];
    n_layer = j["n_layer"];
}

void LLM::load_model(std::string model_path) {
    std::cout << "Loading model weights from " << model_path << std::endl;
    hid_t file_id = H5Fopen((model_path + "/model.h5").c_str(), H5F_ACC_RDONLY, H5P_DEFAULT);
    if (file_id < 0) {
        throw std::runtime_error("Error: Cannot open model.h5");
    }

    std::string base_path = "/model_weights/model";

    // Load embeddings
    std::cout << "> Embeddings" << std::endl;
    std::cout << "  > Allocating host and device memory..." << std::endl;
    h_wte_0.resize(n_vocab * n_embd);
    h_wpe_0.resize(n_ctx * n_embd);
    CHECK_CUDA(hipMalloc(&d_wte_0, n_vocab * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_wpe_0, n_ctx * n_embd * sizeof(float)));
    std::cout << "  > Loading weights..." << std::endl;
    read_dataset(file_id, base_path + "/wte_0", h_wte_0);
    read_dataset(file_id, base_path + "/wpe_0", h_wpe_0);

    // Load layers
    for (int i = 0; i < n_layer; i++) {
        std::cout << "> Layer " << i << std::endl;
        std::cout << "  > Allocating host and device memory..." << std::endl;
        layers.push_back(std::make_unique<Layer>(n_embd, n_head));

        std::string layer_path = base_path + "/h" + std::to_string(i);
        std::cout << "  > Loading weights..." << std::endl;
        layers[i]->load_from_hdf5(file_id, layer_path);
    }

    // Load final layer norm
    std::cout << "> Final layer norm" << std::endl;
    std::cout << "  > Allocating host and device memory..." << std::endl;
    h_ln_f_b_0.resize(n_embd);
    h_ln_f_g_0.resize(n_embd);
    CHECK_CUDA(hipMalloc(&d_ln_f_b_0, n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_ln_f_g_0, n_embd * sizeof(float)));
    std::cout << "  > Loading weights..." << std::endl;
    read_dataset(file_id, base_path + "/ln_f/b_0", h_ln_f_b_0);
    read_dataset(file_id, base_path + "/ln_f/g_0", h_ln_f_g_0);

    // Close the file
    H5Fclose(file_id);

    // Copy weights to device
    copy_params_host_to_device();
}

void LLM::apply_embeddings(int* d_token_ids, float* d_embeddings, int batch_size, int seq_length) {
    // Each thread handles one element (i_batch, i_sequence, i_embedding)
    // in the embedding matrix (batch, sequence, embedding)
    dim3 block_size(32, 32, 1);
    dim3 grid_size((batch_size + block_size.x - 1) / block_size.x,
                   (seq_length + block_size.y - 1) / block_size.y,
                   (n_embd     + block_size.z - 1) / block_size.z);
    embedding_kernel<<<grid_size, block_size>>>(
        d_token_ids, d_wte_0, d_wpe_0, d_embeddings, batch_size, seq_length, n_embd);
}

void LLM::apply_final_layer_norm(float* d_hidden_states, int batch_size, int seq_length) {
    // Each thread handles one token (i_batch, i_sequence, :)
    // in the hidden states (batch, sequence, embedding)
    dim3 block_size(32, 32, 1);
    dim3 grid_size((batch_size + block_size.x - 1) / block_size.x,
                   (seq_length + block_size.y - 1) / block_size.y,
                   1);
    layer_normalization_kernel<<<grid_size, block_size>>>(
        d_hidden_states, d_ln_f_g_0, d_ln_f_b_0, batch_size, seq_length, n_embd);
}

void LLM::apply_lm_head(float* d_hidden_state, float* d_logits, int batch_size, int seq_length) {
    // GPT-2 uses wte as the lm_head
    // Each thread handles one element (i_batch, i_vocab)
    // in the logits (batch, vocab)
    dim3 block_size(32, 32, 1);
    dim3 grid_size((batch_size + block_size.x - 1) / block_size.x,
                   (n_vocab    + block_size.y - 1) / block_size.y,
                   1);
    lm_head_kernel<<<grid_size, block_size>>>(
        d_hidden_state, d_logits, d_wte_0, nullptr, batch_size, n_vocab, n_embd);
}

void LLM::copy_params_host_to_device() {
    CHECK_CUDA(hipMemcpy(d_wte_0, h_wte_0.data(), h_wte_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_wpe_0, h_wpe_0.data(), h_wpe_0.size() * sizeof(float), hipMemcpyHostToDevice));
    for (int i = 0; i < n_layer; i++) {
        layers[i]->copy_host_to_device();
    }
    CHECK_CUDA(hipMemcpy(d_ln_f_b_0, h_ln_f_b_0.data(), h_ln_f_b_0.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_f_g_0, h_ln_f_g_0.data(), h_ln_f_g_0.size() * sizeof(float), hipMemcpyHostToDevice));
}

void LLM::run_interactive() {
    std::cout << "LLM Running Mode. Use CTRL-C to quit.\n";

    while (true) {
        // Get user input
        std::string input;
        std::cout << ">> ";
        std::getline(std::cin, input);

        // Tokenize input
        std::vector<int> h_token_ids = tokenizer.tokenize(input);

        // Print token info
        std::cout << "Token IDs: ";
        for (int id : h_token_ids) {
            std::cout << id << " ";
        }
        std::cout << "\nToken count: " << h_token_ids.size() << std::endl;

        // If empty input, continue
        if (h_token_ids.empty()) {
            std::cout << "Empty input, please try again.\n";
            continue;
        }

        // If input is too long, truncate
        if (h_token_ids.size() > n_ctx) {
            h_token_ids.resize(n_ctx);
            std::cout << "WARNING: Input too long, truncating to " << n_ctx << " tokens." << std::endl;
        }

        // Generate text
        // TODO: Give prior conversation as context
        std::cout << "Generated: ";
        std::vector<int> generated_ids;
        generate_text_recursive(h_token_ids, generated_ids, 1, h_token_ids.size());
        std::cout << std::endl;
    }
}

void LLM::run_inference(const std::vector<std::string>& input_texts) {
    // Tokenize inputs
    std::vector<std::vector<int>> token_batches;
    size_t max_seq_length = 0;
    
    std::cout << "Tokenizing input texts..." << std::endl;
    for (const auto& text : input_texts) {
        std::vector<int> tokens = tokenizer.tokenize(text);
        if (tokens.size() > n_ctx) {
            std::cout << "WARNING: Input too long, truncating to " << n_ctx << " tokens." << std::endl;
            tokens.resize(n_ctx);
        }
        max_seq_length = std::max(max_seq_length, tokens.size());
        token_batches.push_back(std::move(tokens));
    }
    
    // Pad sequences to the same length with EOS token
    for (auto& tokens : token_batches) {
        tokens.resize(max_seq_length, tokenizer.eos_token_id());
    }
    
    size_t batch_size = token_batches.size();
    std::vector<int> h_token_ids(batch_size * max_seq_length);
    
    // Flatten token_batches into h_token_ids
    for (size_t i = 0; i < batch_size; ++i) {
        std::copy(token_batches[i].begin(), token_batches[i].end(), h_token_ids.begin() + i * max_seq_length);
    }
    
    // Run inference
    std::cout << "Running inference on " << batch_size
              << " input texts of max length " << max_seq_length
              << "..." << std::endl;
    std::vector<int> generated_ids;
    generate_text_recursive(h_token_ids, generated_ids, batch_size, max_seq_length);
}


std::vector<float> LLM::forward_pass(const std::vector<int>& token_ids,
                                     int batch_size,
                                     int seq_length) {
    // Allocate device memory for token IDs and embeddings
    int* d_token_ids = nullptr;
    float* d_hidden_states = nullptr;
    float* d_residual = nullptr;
    float* d_temp = nullptr;
    float* d_logits = nullptr;
    std::vector<float> h_logits(n_vocab);

    int token_count = token_ids.size();
    if (token_ids.size() != batch_size * seq_length) {
        throw std::runtime_error("Error: token_ids.size() does not match batch_size * seq_length");
    }

    // Allocate device memory
    CHECK_CUDA(hipMalloc(&d_token_ids, token_count * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_hidden_states, token_count * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_residual, token_count * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_temp, token_count * n_embd * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_logits, batch_size * n_vocab * sizeof(float)));

    // Token IDs
    CHECK_CUDA(hipMemcpy(d_token_ids, token_ids.data(), token_count * sizeof(int), hipMemcpyHostToDevice));

    // Embeddings
    apply_embeddings(d_token_ids, d_hidden_states, batch_size, seq_length);

    // Process through transformer layers
    for (int i = 0; i < n_layer; i++) {
        layers[i]->apply(d_hidden_states, d_residual, d_temp, batch_size, seq_length);
    }

    // Apply final layer norm
    apply_final_layer_norm(d_hidden_states, batch_size, seq_length);

    // Get logits for the last token position
    apply_lm_head(d_hidden_states, d_logits, batch_size, seq_length);

    // Synchronize device
    CHECK_CUDA(hipDeviceSynchronize());

    // Copy logits to host
    CHECK_CUDA(hipMemcpy(h_logits.data(), d_logits, batch_size * n_vocab * sizeof(float), hipMemcpyDeviceToHost));

    // Clean up resources
    clean_up_memory({d_token_ids, d_hidden_states, d_residual, d_logits});

    return h_logits;
}

std::vector<std::pair<float, int>> LLM::get_top_predictions(const std::vector<float>& logits,
                                                            int batch_size,
                                                            int seq_length) {
    std::vector<std::pair<float, int>> probs;
    probs.reserve(batch_size * n_vocab);

    // Scale logits by temperature
    std::vector<float> logits_temp = logits;
    for (auto &logit : logits_temp) {
        logit /= temperature;
    }

    for (int i = 0; i < batch_size; i++) {
        // Apply softmax for each batch
        float max_logit = *std::max_element(logits_temp.begin() + i * n_vocab,
                                            logits_temp.begin() + (i + 1) * n_vocab);
        float sum_exp = 0.0f;
        for (int j = 0; j < n_vocab; j++) {
            float prob = std::exp(logits_temp[i * n_vocab + j] - max_logit);
            sum_exp += prob;
            probs.push_back({prob, j});
        }
        for (int j = 0; j < n_vocab; j++) {
            probs[i * n_vocab + j].first /= sum_exp;
        }

        // Sort by probability (descending)
        std::partial_sort(probs.begin() + i * n_vocab,
                          probs.begin() + i * n_vocab + n_top_predictions,
                          probs.begin() + (i + 1) * n_vocab,
                          [](const auto& a, const auto& b) { return a.first > b.first; });
    }

    return probs;
}

std::vector<int> LLM::sample_tokens(const std::vector<std::pair<float, int>>& probabilities,
                                    int batch_size,
                                    int seq_length) {
    // Sample based on adjusted probabilities
    static std::random_device rd;
    static std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(0.0f, 1.0f);
    std::vector<int> sampled_tokens(batch_size);
    for (int i = 0; i < batch_size; i++) {
        float r = dis(gen);
        float cdf = 0.0f;
        for (int j = 0; j < seq_length; j++) {
            auto& p = probabilities[i * seq_length + j];
            cdf += p.first;
            if (r <= cdf) {
                sampled_tokens[i] = p.second;
                break;
            }
        }
    }
    return sampled_tokens;
}

void LLM::append_new_tokens(std::vector<int>& generated_ids,
                            std::vector<int>& context_ids,
                            const std::vector<int>& new_ids,
                            int batch_size,
                            int seq_length) {
    // Handle generated tokens (accumulating generated tokens)
    // Expand generated tokens
    int seq_length_generated = generated_ids.size() / batch_size;
    generated_ids.resize(batch_size * (seq_length_generated + 1));

    // Copy old generated tokens and add new token at the end
    // Work backwards to avoid overwriting
    for (int i = batch_size - 1; i >= 0; i--) {
        for (int j = seq_length_generated - 1; j >= 0; j--) {
            generated_ids[i * (seq_length_generated + 1) + j] = generated_ids[i * seq_length_generated + j];
        }
        generated_ids[i * (seq_length_generated + 1) + seq_length_generated] = new_ids[i];
    }

    // Handle context tokens (moving window)
    if (seq_length < n_ctx) {
        // Expand context tokens
        context_ids.resize(batch_size * (seq_length + 1));

        // Copy old context tokens and add new token at the end
        // Work backwards to avoid overwriting
        for (int i = batch_size - 1; i >= 0; i--) {
            for (int j = seq_length - 1; j >= 0; j--) {
                context_ids[i * (seq_length + 1) + j] = context_ids[i * seq_length + j];
            }
            context_ids[i * (seq_length + 1) + seq_length] = new_ids[i];
        }
    } else {
        // Shift context tokens to the left and add new token at the end
        for (int i = 0; i < batch_size; i++) {
            for (int j = 0; j < seq_length - 1; j++) {
                context_ids[i * seq_length + j] = context_ids[i * seq_length + j + 1];
            }
            context_ids[i * seq_length + seq_length - 1] = new_ids[i];
        }
    }
}

bool LLM::all_eos(const std::vector<int>& ids,
                  int batch_size,
                  int seq_length) {
    for (int i = 0; i < batch_size; i++) {
        if (ids[i * seq_length + seq_length - 1] != tokenizer.eos_token_id()) {
            return false;
        }
    }
    return true;
}

void LLM::generate_text_recursive(const std::vector<int>& input_ids,
                                  std::vector<int>& generated_ids,
                                  int batch_size,
                                  int seq_length) {
    std::flush(std::cout);
    std::vector<int> context_ids = input_ids;
    
    for (int gen_idx = 0; gen_idx < max_out_length; gen_idx++) {
        // Forward pass for the current sequence
        std::vector<float> logits = forward_pass(context_ids, batch_size, seq_length);
        
        // Get predictions
        std::vector<std::pair<float, int>> probabilities = get_top_predictions(logits, batch_size, seq_length);
        
        // Sample next token
        std::vector<int> next_ids = sample_tokens(probabilities, batch_size, seq_length);
        
        // Add to generated sequence
        append_new_tokens(generated_ids, context_ids, next_ids, batch_size, seq_length);
        
        // Print the token if batch size is 1
        if (batch_size == 1) {
            std::string token_str = tokenizer.detokenize({next_ids[0]});
            std::flush(std::cout);
        }

        // Check for EOS token
        if (all_eos(next_ids, batch_size, seq_length)) {
            break;
        }
    }
}

void LLM::clean_up_memory(const std::vector<void*>& buffers) {
    for (void* buffer : buffers) {
        if (buffer != nullptr) {
            CHECK_CUDA(hipFree(buffer));
        }
    }
}