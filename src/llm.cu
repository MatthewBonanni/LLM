#include "hip/hip_runtime.h"
#include "llm.cuh"

#include <hip/hip_runtime.h>

#include <fstream>
#include <iostream>
#include <stdexcept>
#include <string>
#include <vector>
#include <random>

#include <nlohmann/json.hpp>
#include <hdf5.h>
#include <hdf5_hl.h>

#include "utils.cuh"
#include "io.cuh"
#include "tokenizer.cuh"
#include "layer.cuh"
#include "kernels.cuh"

LLM::LLM(const std::string& model_path) :
        tokenizer(model_path),
        d_wte_0(nullptr),
        d_wpe_0(nullptr),
        d_ln_f_b_0(nullptr),
        d_ln_f_g_0(nullptr),
        max_out_length(50),
        temperature(0.7f),
        n_top_predictions(200) {
    load_hparams(model_path);
    load_model(model_path);

    if (n_top_predictions > n_vocab) {
        n_top_predictions = n_vocab;
    }
}

LLM::~LLM() {}

void LLM::print() {
    printf("--------------------------------\n");
    printf("LLM Configuration\n");
    printf("--------------------------------\n");
    printf("Model hyperparameters:\n");
    printf("> n_vocab: %d\n", n_vocab);
    printf("> n_ctx: %d\n", n_ctx);
    printf("> n_embd: %d\n", n_embd);
    printf("> n_head: %d\n", n_head);
    printf("> n_layer: %d\n", n_layer);
    printf("Runtime parameters:\n");
    printf("> max_out_length: %d\n", max_out_length);
    printf("> temperature: %.2f\n", temperature);
    printf("> n_top_predictions: %d\n", n_top_predictions);
    printf("--------------------------------\n");
}

void LLM::load_hparams(std::string model_path) {
    std::ifstream file(model_path + "/hparams.json");
    if (!file) {
        std::cerr << "Error: Cannot open encoder.json" << std::endl;
        return;
    }

    nlohmann::json j;
    file >> j;
    n_vocab = j["n_vocab"];
    n_ctx = j["n_ctx"];
    n_embd = j["n_embd"];
    n_head = j["n_head"];
    n_layer = j["n_layer"];
}

void LLM::load_model(std::string model_path) {
    std::cout << "Loading model weights from " << model_path << std::endl;
    hid_t file_id = H5Fopen((model_path + "/model.h5").c_str(), H5F_ACC_RDONLY, H5P_DEFAULT);
    if (file_id < 0) {
        throw std::runtime_error("Error: Cannot open model.h5");
    }

    std::string base_path = "/model_weights/model";

    // Load embeddings
    std::cout << "> Embeddings" << std::endl;
    std::cout << "  > Allocating host and device memory..." << std::endl;
    h_wte_0.resize(n_vocab * n_embd);
    h_wpe_0.resize(n_ctx * n_embd);
    CHECK_CUDA(hipMalloc(&d_wte_0, n_vocab * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_wpe_0, n_ctx * n_embd * sizeof(fp_t)));
    std::cout << "  > Loading weights..." << std::endl;
    read_dataset(file_id, base_path + "/wte_0", h_wte_0);
    read_dataset(file_id, base_path + "/wpe_0", h_wpe_0);

    // Load layers
    for (uint64_t i = 0; i < n_layer; i++) {
        std::cout << "> Layer " << i << std::endl;
        std::cout << "  > Allocating host and device memory..." << std::endl;
        layers.push_back(std::make_unique<Layer>(n_embd, n_head));

        std::string layer_path = base_path + "/h" + std::to_string(i);
        std::cout << "  > Loading weights..." << std::endl;
        layers[i]->load_from_hdf5(file_id, layer_path);
    }

    // Load final layer norm
    std::cout << "> Final layer norm" << std::endl;
    std::cout << "  > Allocating host and device memory..." << std::endl;
    h_ln_f_b_0.resize(n_embd);
    h_ln_f_g_0.resize(n_embd);
    CHECK_CUDA(hipMalloc(&d_ln_f_b_0, n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_ln_f_g_0, n_embd * sizeof(fp_t)));
    std::cout << "  > Loading weights..." << std::endl;
    read_dataset(file_id, base_path + "/ln_f/b_0", h_ln_f_b_0);
    read_dataset(file_id, base_path + "/ln_f/g_0", h_ln_f_g_0);

    // Close the file
    H5Fclose(file_id);

    // Copy weights to device
    copy_params_host_to_device();
}

void LLM::apply_embeddings(id_t* d_token_ids, fp_t* d_embeddings, uint64_t batch_size, uint64_t seq_length) {
    // Each thread handles one token (i_batch, i_sequence, :)
    // in the token_ids (batch, sequence, embedding)
    dim3 block_size(32, 32, 1);
    dim3 grid_size((batch_size + block_size.x - 1) / block_size.x,
                   (seq_length + block_size.y - 1) / block_size.y,
                   1);
    embedding_kernel<<<grid_size, block_size>>>(
        d_token_ids, d_wte_0, d_wpe_0, d_embeddings,
        batch_size, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());
}

void LLM::apply_final_layer_norm(fp_t* d_hidden_states, uint64_t batch_size, uint64_t seq_length) {
    // Each thread handles one token (i_batch, i_sequence, :)
    // in the hidden states (batch, sequence, embedding)
    dim3 block_size(32, 32, 1);
    dim3 grid_size((batch_size + block_size.x - 1) / block_size.x,
                   (seq_length + block_size.y - 1) / block_size.y,
                   1);
    layer_normalization_kernel<<<grid_size, block_size>>>(
        d_hidden_states, d_ln_f_g_0, d_ln_f_b_0,
        batch_size, seq_length, n_embd);
    CHECK_CUDA(hipGetLastError());
}

void LLM::apply_lm_head(fp_t* d_hidden_state, fp_t* d_logits, uint64_t batch_size, uint64_t seq_length) {
    // GPT-2 uses wte as the lm_head
    // Each thread handles one element (i_batch, i_vocab)
    // in the logits (batch, vocab)
    dim3 block_size(32, 32, 1);
    dim3 grid_size((batch_size + block_size.x - 1) / block_size.x,
                   (n_vocab    + block_size.y - 1) / block_size.y,
                   1);
    lm_head_kernel<<<grid_size, block_size>>>(
        d_hidden_state, d_logits, d_wte_0, nullptr,
        batch_size, seq_length, n_vocab, n_embd);
    CHECK_CUDA(hipGetLastError());
}

void LLM::copy_params_host_to_device() {
    CHECK_CUDA(hipMemcpy(d_wte_0, h_wte_0.data(), h_wte_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_wpe_0, h_wpe_0.data(), h_wpe_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    for (uint64_t i = 0; i < n_layer; i++) {
        layers[i]->copy_host_to_device();
    }
    CHECK_CUDA(hipMemcpy(d_ln_f_b_0, h_ln_f_b_0.data(), h_ln_f_b_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_ln_f_g_0, h_ln_f_g_0.data(), h_ln_f_g_0.size() * sizeof(fp_t), hipMemcpyHostToDevice));
}

void LLM::run_interactive() {
    std::cout << "LLM Running Mode. Use CTRL-C to quit.\n";

    while (true) {
        // Get user input
        std::string input;
        std::cout << ">> ";
        std::getline(std::cin, input);

        // Tokenize input
        std::vector<id_t> h_token_ids = tokenizer.tokenize(input);

        // Print token info
        std::cout << "Token IDs: ";
        for (auto id : h_token_ids) {
            std::cout << id << " ";
        }
        std::cout << "\nToken count: " << h_token_ids.size() << std::endl;

        // If empty input, continue
        if (h_token_ids.empty()) {
            std::cout << "Empty input, please try again.\n";
            continue;
        }

        // If input is too long, truncate
        if (h_token_ids.size() > n_ctx) {
            h_token_ids.resize(n_ctx);
            std::cout << "WARNING: Input too long, truncating to " << n_ctx << " tokens." << std::endl;
        }

        // Generate text
        // TODO: Give prior conversation as context
        std::cout << "Generated: ";
        std::vector<id_t> generated_ids;
        uint64_t seq_length = h_token_ids.size();
        generate_text_recursive(h_token_ids, generated_ids, 1, seq_length);

        // Print generated tokens
        for (auto id : generated_ids) {
            std::string token_str = tokenizer.detokenize({id});
            std::cout << token_str << " ";
        }
        std::cout << std::endl;
    }
}

void LLM::tokenize(const std::vector<std::string>& input_texts,
                   std::vector<id_t>& token_ids,
                   uint64_t& batch_size,
                   uint64_t& seq_length) {
    // Tokenize inputs
    std::vector<std::vector<id_t>> token_ids_batches;
    seq_length = 0;

    std::cout << "Tokenizing input texts..." << std::endl;
    for (const auto& text : input_texts) {
        std::vector<id_t> token_ids_i = tokenizer.tokenize(text);
        if (token_ids_i.size() > n_ctx) {
            std::cout << "WARNING: Input too long, truncating to " << n_ctx << " tokens." << std::endl;
            token_ids_i.resize(n_ctx);
        }
        seq_length = std::max(seq_length, (uint64_t)token_ids_i.size());
        token_ids_batches.push_back(std::move(token_ids_i));
    }

    // Pad sequences to the same length with EOS token
    for (auto& token_ids_i : token_ids_batches) {
        token_ids_i.resize(seq_length, tokenizer.eos_token_id());
    }

    batch_size = token_ids_batches.size();
    token_ids.resize(batch_size * seq_length);

    // Flatten token_batches into token_ids
    for (uint64_t i = 0; i < batch_size; ++i) {
        std::copy(token_ids_batches[i].begin(),
                  token_ids_batches[i].end(),
                  token_ids.begin() + i * seq_length);
    }
}

void LLM::write_token_ids(const std::string& h5_file_path,
                          const std::vector<id_t>& token_ids,
                          uint64_t batch_size,
                          uint64_t seq_length) {
    // Write token IDs to H5 file
    hid_t file_id = H5Fcreate(h5_file_path.c_str(), H5F_ACC_TRUNC, H5P_DEFAULT, H5P_DEFAULT);
    if (file_id < 0) {
        throw std::runtime_error("Error: Cannot create H5 file");
    }
    
    // Create dataset for token IDs
    hsize_t dims[1] = {token_ids.size()};
    hid_t dataspace_id = H5Screate_simple(1, dims, nullptr);
    hid_t dataset_id = H5Dcreate(file_id, "/token_ids", H5T_NATIVE_INT, dataspace_id, H5P_DEFAULT, H5P_DEFAULT, H5P_DEFAULT);
    
    // Write token IDs data
    H5Dwrite(dataset_id, H5T_NATIVE_INT, H5S_ALL, H5S_ALL, H5P_DEFAULT, token_ids.data());
    
    // Create attribute space (scalar)
    hid_t attr_space = H5Screate(H5S_SCALAR);
    
    // Create and write batch_size attribute
    hid_t attr_id_batch = H5Acreate(dataset_id, "batch_size", H5T_NATIVE_INT, attr_space, H5P_DEFAULT, H5P_DEFAULT);
    H5Awrite(attr_id_batch, H5T_NATIVE_INT, &batch_size);
    H5Aclose(attr_id_batch);
    
    // Create and write seq_length attribute
    hid_t attr_id_seq = H5Acreate(dataset_id, "seq_length", H5T_NATIVE_INT, attr_space, H5P_DEFAULT, H5P_DEFAULT);
    H5Awrite(attr_id_seq, H5T_NATIVE_INT, &seq_length);
    H5Aclose(attr_id_seq);
    
    // Close resources
    H5Sclose(attr_space);
    H5Dclose(dataset_id);
    H5Sclose(dataspace_id);
    H5Fclose(file_id);
}

void LLM::load_token_ids(const std::string& h5_file_path,
                         std::vector<id_t>& token_ids,
                         uint64_t& batch_size,
                         uint64_t& seq_length) {
    // Load token IDs from H5 file
    hid_t file_id = H5Fopen(h5_file_path.c_str(), H5F_ACC_RDONLY, H5P_DEFAULT);
    if (file_id < 0) {
        throw std::runtime_error("Error: Cannot open H5 file");
    }
    
    // Open the dataset
    hid_t dataset_id = H5Dopen(file_id, "/token_ids", H5P_DEFAULT);
    if (dataset_id < 0) {
        H5Fclose(file_id);
        throw std::runtime_error("Error: Cannot open dataset");
    }
    
    // Get the dataspace to determine the size
    hid_t dataspace_id = H5Dget_space(dataset_id);
    hsize_t dims[1];
    H5Sget_simple_extent_dims(dataspace_id, dims, NULL);
    
    // Resize the vector to hold the data
    token_ids.resize(dims[0]);
    
    // Read token IDs data
    H5Dread(dataset_id, H5T_NATIVE_INT, H5S_ALL, H5S_ALL, H5P_DEFAULT, token_ids.data());
    
    // Read batch_size attribute
    int32_t batch_size_attr;
    hid_t attr_id_batch = H5Aopen(dataset_id, "batch_size", H5P_DEFAULT);
    H5Aread(attr_id_batch, H5T_NATIVE_INT, &batch_size_attr);
    H5Aclose(attr_id_batch);
    batch_size = batch_size_attr;

    // Read seq_length attribute
    int32_t seq_length_attr;
    hid_t attr_id_seq = H5Aopen(dataset_id, "seq_length", H5P_DEFAULT);
    H5Aread(attr_id_seq, H5T_NATIVE_INT, &seq_length_attr);
    H5Aclose(attr_id_seq);
    seq_length = seq_length_attr;

    // Close resources
    H5Sclose(dataspace_id);
    H5Dclose(dataset_id);
    H5Fclose(file_id);
}

void LLM::tokenize_write_and_run_inference(const std::vector<std::string>& input_texts) {
    // Tokenize input
    uint64_t batch_size, seq_length;
    std::vector<id_t> token_ids;
    tokenize(input_texts, token_ids, batch_size, seq_length);

    // Write token IDs to H5 file
    std::string h5_file_path = "token_ids.h5";
    write_token_ids(h5_file_path, token_ids, batch_size, seq_length);

    // Run inference
    run_inference(token_ids, batch_size, seq_length);
}

void LLM::load_tokens_and_run_inference(const std::string& h5_file_path) {
    // Load token IDs from H5 file
    uint64_t batch_size, seq_length;
    std::vector<id_t> token_ids;
    load_token_ids(h5_file_path, token_ids, batch_size, seq_length);

    // Run inference
    run_inference(token_ids, batch_size, seq_length);
}

void LLM::run_inference(const std::vector<id_t>& token_ids,
                        uint64_t batch_size,
                        uint64_t seq_length) {
    std::cout << "Running inference on " << batch_size
              << " input sequences of max length " << seq_length
              << "..." << std::endl;
    std::vector<id_t> generated_ids;
    generate_text_recursive(token_ids, generated_ids, batch_size, seq_length);
}


std::vector<fp_t> LLM::forward_pass(const std::vector<id_t>& token_ids,
                                    uint64_t batch_size,
                                    uint64_t seq_length) {
    // Allocate device memory for token IDs and embeddings
    id_t* d_token_ids = nullptr;
    fp_t* d_hidden_states = nullptr;
    fp_t* d_residual = nullptr;
    fp_t* d_temp = nullptr;
    fp_t* d_logits = nullptr;
    std::vector<fp_t> h_logits(batch_size * n_vocab);

    uint64_t token_count = batch_size * seq_length;
    if (token_ids.size() != token_count) {
        throw std::runtime_error("Error: token_ids.size() does not match batch_size * seq_length");
    }

    // Allocate device memory
    std::cout << "HERE: ALLOCATING DEVICE MEMORY" << std::endl;
    uint64_t free_memory, total_memory;
    hipMemGetInfo(&free_memory, &total_memory);
    std::cout << "Total GPU memory: " << total_memory / (1024 * 1024) << " MB" << std::endl;
    std::cout << "Free GPU memory: " << free_memory / (1024 * 1024) << " MB" << std::endl;
    uint64_t required_memory = token_count * sizeof(id_t) +
                               token_count * n_embd * sizeof(fp_t) * 3 +
                               batch_size * n_vocab * sizeof(fp_t);
    std::cout << "Required GPU memory: " << required_memory / (1024 * 1024) << " MB" << std::endl;
    if (required_memory > free_memory) {
        throw std::runtime_error("Not enough GPU memory available for allocation.");
    }
    CHECK_CUDA(hipMalloc(&d_token_ids,     token_count * sizeof(id_t)));
    CHECK_CUDA(hipMalloc(&d_hidden_states, token_count * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_residual,      token_count * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_temp,          token_count * n_embd * sizeof(fp_t)));
    CHECK_CUDA(hipMalloc(&d_logits,        batch_size * n_vocab * sizeof(fp_t)));

    // Token IDs
    CHECK_CUDA(hipMemcpy(d_token_ids, token_ids.data(), token_count * sizeof(id_t), hipMemcpyHostToDevice));

    // Embeddings
    apply_embeddings(d_token_ids, d_hidden_states, batch_size, seq_length);

    // Process through transformer layers
    for (uint64_t i = 0; i < n_layer; i++) {
        layers[i]->apply(d_hidden_states, d_residual, d_temp, batch_size, seq_length);
    }

    // Apply final layer norm
    apply_final_layer_norm(d_hidden_states, batch_size, seq_length);

    // Get logits for the last token position
    apply_lm_head(d_hidden_states, d_logits, batch_size, seq_length);

    // Synchronize device
    CHECK_CUDA(hipDeviceSynchronize());

    // Copy logits to host
    CHECK_CUDA(hipMemcpy(h_logits.data(), d_logits, batch_size * n_vocab * sizeof(fp_t), hipMemcpyDeviceToHost));

    // Clean up resources
    clean_up_memory({d_token_ids, d_hidden_states, d_residual, d_logits});

    return h_logits;
}

std::vector<std::pair<fp_t, id_t>> LLM::get_top_predictions(const std::vector<fp_t>& logits,
                                                            uint64_t batch_size) {
    std::vector<std::pair<fp_t, id_t>> probs;
    probs.reserve(batch_size * n_vocab);

    // Scale logits by temperature
    std::vector<fp_t> logits_temp = logits;
    for (auto &logit : logits_temp) {
        logit /= temperature;
    }

    for (uint64_t i = 0; i < batch_size; i++) {
        // Apply softmax for each batch
        fp_t max_logit = *std::max_element(logits_temp.begin() + i * n_vocab,
                                            logits_temp.begin() + (i + 1) * n_vocab);
        fp_t sum_exp = 0.0f;
        for (uint64_t j = 0; j < n_vocab; j++) {
            fp_t prob = std::exp(logits_temp[i * n_vocab + j] - max_logit);
            sum_exp += prob;
            probs.push_back({prob, j});
        }
        for (uint64_t j = 0; j < n_vocab; j++) {
            probs[i * n_vocab + j].first /= sum_exp;
        }

        // Sort by probability (descending)
        std::partial_sort(probs.begin() + i * n_vocab,
                          probs.begin() + i * n_vocab + n_top_predictions,
                          probs.begin() + (i + 1) * n_vocab,
                          [](const auto& a, const auto& b) { return a.first > b.first; });
    }

    return probs;
}

std::vector<id_t> LLM::sample_tokens(const std::vector<std::pair<fp_t, id_t>>& probabilities,
                                     uint64_t batch_size) {
    // Sample based on adjusted probabilities
    static std::random_device rd;
    static std::mt19937 gen(rd());
    std::uniform_real_distribution<fp_t> dis(0.0f, 1.0f);
    std::vector<id_t> sampled_tokens(batch_size);
    for (uint64_t i = 0; i < batch_size; i++) {
        fp_t r = dis(gen);
        fp_t cdf = 0.0f;
        for (uint64_t j = 0; j < n_vocab; j++) {
            auto& p = probabilities[i * n_vocab + j];
            cdf += p.first;
            if (r <= cdf) {
                sampled_tokens[i] = p.second;
                break;
            }
        }
    }
    return sampled_tokens;
}

void LLM::append_new_tokens(std::vector<id_t>& generated_ids,
                            std::vector<id_t>& context_ids,
                            const std::vector<id_t>& new_ids,
                            uint64_t batch_size,
                            uint64_t& seq_length) {
    // Handle generated tokens (accumulating generated tokens)
    // Expand generated tokens
    uint64_t seq_length_generated = generated_ids.size() / batch_size;
    generated_ids.resize(batch_size * (seq_length_generated + 1));

    // Copy old generated tokens and add new token at the end
    // Work backwards to avoid overwriting
    for (int64_t i = batch_size - 1; i >= 0; i--) {
        for (int64_t j = seq_length_generated - 1; j >= 0; j--) {
            generated_ids[i * (seq_length_generated + 1) + j] = generated_ids[i * seq_length_generated + j];
        }
        generated_ids[i * (seq_length_generated + 1) + seq_length_generated] = new_ids[i];
    }

    // Handle context tokens (moving window)
    if (seq_length < n_ctx) {
        // Expand context tokens
        context_ids.resize(batch_size * (seq_length + 1));

        // Copy old context tokens and add new token at the end
        // Work backwards to avoid overwriting
        for (int64_t i = batch_size - 1; i >= 0; i--) {
            for (int64_t j = seq_length - 1; j >= 0; j--) {
                context_ids[i * (seq_length + 1) + j] = context_ids[i * seq_length + j];
            }
            context_ids[i * (seq_length + 1) + seq_length] = new_ids[i];
        }

        seq_length++;
    } else {
        // Shift context tokens to the left and add new token at the end
        for (int64_t i = 0; i < batch_size; i++) {
            for (int64_t j = 0; j < seq_length - 1; j++) {
                context_ids[i * seq_length + j] = context_ids[i * seq_length + j + 1];
            }
            context_ids[i * seq_length + seq_length - 1] = new_ids[i];
        }
    }
}

bool LLM::all_eos(const std::vector<id_t>& ids,
                  uint64_t batch_size,
                  uint64_t seq_length) {
    for (uint64_t i = 0; i < batch_size; i++) {
        if (ids[i * seq_length + seq_length - 1] != tokenizer.eos_token_id()) {
            return false;
        }
    }
    return true;
}

void LLM::generate_text_recursive(const std::vector<id_t>& input_ids,
                                  std::vector<id_t>& generated_ids,
                                  uint64_t batch_size,
                                  uint64_t& seq_length) {
    std::flush(std::cout);
    std::vector<id_t> context_ids = input_ids;
    
    for (uint64_t gen_idx = 0; gen_idx < max_out_length; gen_idx++) {
        // Forward pass for the current sequence
        std::vector<fp_t> logits = forward_pass(context_ids, batch_size, seq_length);
        
        // Get predictions
        std::vector<std::pair<fp_t, id_t>> probabilities = get_top_predictions(logits, batch_size);
        
        // Sample next token
        std::vector<id_t> next_ids = sample_tokens(probabilities, batch_size);
        
        // Add to generated sequence
        append_new_tokens(generated_ids, context_ids, next_ids, batch_size, seq_length);
        
        // Print the token if batch size is 1
        if (batch_size == 1) {
            std::string token_str = tokenizer.detokenize({next_ids[0]});
            std::cout << token_str;
            std::flush(std::cout);
        } else {
            std::cout << "Finished generating " << gen_idx + 1 << " tokens." << std::endl;
        }

        // Check for EOS token
        if (all_eos(next_ids, batch_size, seq_length)) {
            break;
        }
    }
}

void LLM::clean_up_memory(const std::vector<void*>& buffers) {
    for (void* buffer : buffers) {
        if (buffer != nullptr) {
            CHECK_CUDA(hipFree(buffer));
        }
    }
}