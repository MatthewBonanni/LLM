#include "utils.cuh"

#include <hip/hip_runtime.h>

void clean_up_memory(std::vector<void*>& buffers) {
    for (void*& buffer : buffers) {
        if (buffer != nullptr) {
            CHECK_CUDA(hipFree(buffer));
        }
        buffer = nullptr;
    }
}